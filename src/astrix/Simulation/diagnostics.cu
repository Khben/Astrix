#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file diagnostics.cu
\brief Calculating various diagnostics for Simulations

\section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/
#include <iostream>
#include <fstream>

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "../Mesh/mesh.h"
#include "./simulation.h"
#include "../Common/cudaLow.h"
#include "../Common/inlineMath.h"
#include "./Param/simulationparameter.h"

namespace astrix {

//##############################################################################
//##############################################################################

__host__ __device__
void KineticEnergySingle(unsigned int i, const real *pVarea,
                         real4 *pState, real *Ex, real *Ey)
{
  real half = (real) 0.5;

  real d = pState[i].x;
  real m = pState[i].y;
  real n = pState[i].z;

  Ex[i] = pVarea[i]*half*m*m/d;
  Ey[i] = pVarea[i]*half*n*n/d;
}

__host__ __device__
void KineticEnergySingle(unsigned int i, const real *pVarea,
                         real3 *pState, real *Ex, real *Ey)
{
  real half = (real) 0.5;

  real d = pState[i].x;
  real m = pState[i].y;
  real n = pState[i].z;

  Ex[i] = pVarea[i]*half*m*m/d;
  Ey[i] = pVarea[i]*half*n*n/d;
}

  __host__ __device__
void KineticEnergySingle(unsigned int i, const real *pVarea,
                         real *pState, real *Ex, real *Ey)
{
  Ex[i] = (real) 0.0;
  Ey[i] = (real) 0.0;
}

//######################################################################
//######################################################################

__global__ void
devKineticEnergy(unsigned int nVertex, const real *pVarea,
                 realNeq *pState, real *Ex, real *Ey)
{
  // n = vertex number
  unsigned int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nVertex) {
    KineticEnergySingle(n, pVarea, pState, Ex, Ey);

    n += blockDim.x*gridDim.x;
  }
}

//######################################################################
//######################################################################

real2 Simulation::KineticEnergy()
{
  unsigned int nVertex = mesh->GetNVertex();

  realNeq *pState = vertexState->GetPointer();

  const real *pVarea = mesh->VertexAreaData();

  Array<real> *Ex = new Array<real>(1, cudaFlag, nVertex);
  Array<real> *Ey = new Array<real>(1, cudaFlag, nVertex);
  real *pEx = Ex->GetPointer();
  real *pEy = Ey->GetPointer();

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devKineticEnergy,
                                       (size_t) 0, 0);

    devKineticEnergy<<<nBlocks, nThreads>>>
      (nVertex, pVarea, pState, pEx, pEy);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (unsigned int n = 0; n < nVertex; n++)
      KineticEnergySingle(n, pVarea, pState, pEx, pEy);
  }

  real ex = Ex->Sum();
  real ey = Ey->Sum();

  delete Ex;
  delete Ey;

  real2 ret;
  ret.x = ex;
  ret.y = ey;

  return ret;
}

//##############################################################################
//##############################################################################

__host__ __device__
void ThermalEnergySingle(unsigned int i, const real *pVarea,
                         real4 *pState, real *E)
{
  real half = (real) 0.5;

  real d = pState[i].x;
  real m = pState[i].y;
  real n = pState[i].z;
  real e = pState[i].w;

  E[i] = pVarea[i]*(e - half*(m*m + n*n)/d);
}

__host__ __device__
void ThermalEnergySingle(unsigned int i, const real *pVarea,
                         real3 *pState, real *E)
{
  E[i] = (real) 0.0;
}

__host__ __device__
void ThermalEnergySingle(unsigned int i, const real *pVarea,
                         real *pState, real *E)
{
  E[i] = (real) 0.0;
}

//######################################################################
//######################################################################

__global__ void
devThermalEnergy(unsigned int nVertex, const real *pVarea,
                 realNeq *pState, real *E)
{
  // n = vertex number
  unsigned int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nVertex) {
    ThermalEnergySingle(n, pVarea, pState, E);

    n += blockDim.x*gridDim.x;
  }
}

//######################################################################
//######################################################################

real Simulation::ThermalEnergy()
{
  unsigned int nVertex = mesh->GetNVertex();

  realNeq *pState = vertexState->GetPointer();

  const real *pVarea = mesh->VertexAreaData();

  Array<real> *E = new Array<real>(1, cudaFlag, nVertex);
  real *pE = E->GetPointer();

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devKineticEnergy,
                                       (size_t) 0, 0);

    devThermalEnergy<<<nBlocks, nThreads>>>
      (nVertex, pVarea, pState, pE);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (unsigned int n = 0; n < nVertex; n++)
      ThermalEnergySingle(n, pVarea, pState, pE);
  }

  real e = E->Sum();

  delete E;

  return e;
}

//##############################################################################
//##############################################################################

__host__ __device__
void PotentialEnergySingle(unsigned int i, const real2 *pVc, const real *pVarea,
                           real4 *pState, real *E, ProblemDefinition problemDef)
{
  real d = pState[i].x;
  real vertY = pVc[i].y;

  real pot = 0.0;
  if (problemDef == PROBLEM_SOURCE)
    pot = 0.1*vertY;

  E[i] = pVarea[i]*d*pot;
}

//######################################################################
//######################################################################

__global__ void
devPotentialEnergy(unsigned int nVertex, const real2 *pVc, const real *pVarea,
                   realNeq *pState, real *E, ProblemDefinition problemDef)
{
  // n = vertex number
  unsigned int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nVertex) {
    PotentialEnergySingle(n, pVc, pVarea, pState, E, problemDef);

    n += blockDim.x*gridDim.x;
  }
}

//######################################################################
//######################################################################

real Simulation::PotentialEnergy()
{
  unsigned int nVertex = mesh->GetNVertex();

  realNeq *pState = vertexState->GetPointer();

  const real *pVarea = mesh->VertexAreaData();
  const real2 *pVc = mesh->VertexCoordinatesData();

  Array<real> *E = new Array<real>(1, cudaFlag, nVertex);
  real *pE = E->GetPointer();

  ProblemDefinition p = simulationParameter->problemDef;

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devPotentialEnergy,
                                       (size_t) 0, 0);

    devPotentialEnergy<<<nBlocks, nThreads>>>
      (nVertex, pVc, pVarea, pState, pE, p);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (unsigned int n = 0; n < nVertex; n++)
      PotentialEnergySingle(n, pVc, pVarea, pState, pE, p);
  }

  real e = E->Sum();

  delete E;

  return e;
}

//##############################################################################
//##############################################################################

__host__ __device__
void DensityErrorSingle(unsigned int i, const real *pVarea,
                        const real2 *pVc,
                        real4 *pState, real4 *pStateOld, real *E)
{
  real d = pState[i].x;
  real d0 = pStateOld[i].x;

  E[i] = pVarea[i]*std::abs(d - d0);
}

__host__ __device__
void DensityErrorSingle(unsigned int i, const real *pVarea,
                        const real2 *pVc,
                        real3 *pState, real3 *pStateOld, real *E)
{
  real d = pState[i].x;
  real d0 = pStateOld[i].x;

  E[i] = pVarea[i]*std::abs(d - d0);
}

__host__ __device__
void DensityErrorSingle(unsigned int i, const real *pVarea,
                        const real2 *pVc,
                        real *pState, real *pStateOld, real *E)
{
  real d = pState[i];
  real d0 = pStateOld[i];

  E[i] = pVarea[i]*std::abs(d - d0);
}

//######################################################################
//######################################################################

__global__ void
devDensityError(unsigned int nVertex, const real *pVarea,
                const real2 *pVc,
                realNeq *pState, realNeq *pStateOld, real *E)
{
  // n = vertex number
  unsigned int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nVertex) {
    DensityErrorSingle(n, pVarea, pVc, pState, pStateOld, E);

    n += blockDim.x*gridDim.x;
  }
}

//######################################################################
//######################################################################

real Simulation::DensityError()
{
  real2 Ekin = KineticEnergy();
  real Eth = ThermalEnergy();

  vertexStateOld->SetEqual(vertexState);

  // Assume this gets the correct solution into vertexState
  SetInitial(simulationTime);

  unsigned int nVertex = mesh->GetNVertex();

  realNeq *pState = vertexState->GetPointer();
  realNeq *pStateOld = vertexStateOld->GetPointer();

  const real *pVarea = mesh->VertexAreaData();
  const real2 *pVc = mesh->VertexCoordinatesData();

  Array<real> *E = new Array<real>(1, cudaFlag, nVertex);
  real *pE = E->GetPointer();

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devDensityError,
                                       (size_t) 0, 0);

    devDensityError<<<nBlocks, nThreads>>>
      (nVertex, pVarea, pVc, pState, pStateOld, pE);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (unsigned int n = 0; n < nVertex; n++)
      DensityErrorSingle(n, pVarea, pVc, pState, pStateOld, pE);
  }

  real e = E->Sum()/mesh->GetTotalArea();

  delete E;

  vertexState->SetEqual(vertexStateOld);

  return e;
}

}  // namespace astrix
