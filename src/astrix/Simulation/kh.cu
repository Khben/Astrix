#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file kh.cu
\brief Diagnostics for Kelvin-Helmholtz test problem

\section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/
#include <iostream>
#include <fstream>

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "../Mesh/mesh.h"
#include "./simulation.h"
#include "../Common/cudaLow.h"
#include "../Common/inlineMath.h"
#include "./Param/simulationparameter.h"

namespace astrix {

//##############################################################################
//##############################################################################

__host__ __device__
void AddEigenVectorSingle(unsigned int i, const real2 *pVc, real4 *pState,
                          real *dR, real*dI,
                          real *uR, real *uI,
                          real *vR, real *vI,
                          real dyKH, real kxKH, real *yKH,
                          real miny, real maxy, real G, real G1)
{
  real x = pVc[i].x;
  real y = pVc[i].y;

  if (y < miny) y += (maxy - miny);
  if (y > maxy) y -= (maxy - miny);

  int jj = (int)((y - yKH[0])/dyKH);
#ifndef __CUDA_ARCH__
  if (jj < 0 || jj > 128) {
    std::cout << jj << " " << y << " " << yKH[0] << " " << yKH[129]
              << std::endl;
    int qq; std::cin >> qq;
  }
#endif

  real dRj = dR[jj] + (y - yKH[jj])*(dR[jj + 1] - dR[jj])/dyKH;
  real dIj = dI[jj] + (y - yKH[jj])*(dI[jj + 1] - dI[jj])/dyKH;
  real uRj = uR[jj] + (y - yKH[jj])*(uR[jj + 1] - uR[jj])/dyKH;
  real uIj = uI[jj] + (y - yKH[jj])*(uI[jj + 1] - uI[jj])/dyKH;
  real vRj = vR[jj] + (y - yKH[jj])*(vR[jj + 1] - vR[jj])/dyKH;
  real vIj = vI[jj] + (y - yKH[jj])*(vI[jj + 1] - vI[jj])/dyKH;

  real d0 = pState[i].x;
  real a0 = pState[i].y;
  real b0 = pState[i].z;
  real e0 = pState[i].w;
  real p0 = G1*(e0 - 0.5*(a0*a0 + b0*b0)/d0);

  pState[i].x = d0 + dRj*cos(2.0*M_PI*kxKH*x) - dIj*sin(2.0*M_PI*kxKH*x);
  pState[i].y = a0 + d0*uRj*cos(2.0*M_PI*kxKH*x) - d0*uIj*sin(2.0*M_PI*kxKH*x);
  pState[i].z = b0 + d0*vRj*cos(2.0*M_PI*kxKH*x) - d0*vIj*sin(2.0*M_PI*kxKH*x);
  real pr = p0 + G*p0*(dRj*cos(2.0*M_PI*kxKH*x) - dIj*sin(2.0*M_PI*kxKH*x))/d0;
  pState[i].w = 0.5*(Sq(pState[i].y) + Sq(pState[i].z))/pState[i].x + pr/G1;
}

__host__ __device__
void AddEigenVectorSingle(unsigned int i, const real2 *pVc, real3 *pState,
                          real *dR, real*dI,
                          real *uR, real *uI,
                          real *vR, real *vI,
                          real dyKH, real kxKH, real *yKH,
                          real miny, real maxy, real G, real G1)
{
  // Dummy function; no eigenvector to add if solving isothermal equation
}

__host__ __device__
void AddEigenVectorSingle(unsigned int i, const real2 *pVc, real *pState,
                          real *dR, real*dI,
                          real *uR, real *uI,
                          real *vR, real *vI,
                          real dyKH, real kxKH, real *yKH,
                          real miny, real maxy, real G, real G1)
{
  // Dummy function; no eigenvector to add if solving scalar equation
}

//######################################################################
//######################################################################

template<class realNeq, ConservationLaw CL>
__global__ void
devAddEigenVector(unsigned int nVertex, const real2 *pVc, realNeq *pState,
                  real *dR, real*dI, real *uR, real *uI, real *vR, real *vI,
                  real dyKH, real kxKH, real *yKH,
                  real miny, real maxy, real G, real G1)
{
  // n = vertex number
  unsigned int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nVertex) {
    AddEigenVectorSingle(n, pVc, pState, dR, dI, uR, uI, vR, vI,
                         dyKH, kxKH, yKH, miny, maxy, G, G1);

    n += blockDim.x*gridDim.x;
  }
}

//######################################################################
//######################################################################

template <class realNeq, ConservationLaw CL>
void Simulation<realNeq, CL>::KHAddEigenVector()
{
  unsigned int nVertex = mesh->GetNVertex();

  realNeq *pState = vertexState->GetPointer();
  real G = simulationParameter->specificHeatRatio;

  const real2 *pVc = mesh->VertexCoordinatesData();

  // Read in KH eigenvector
  std::ifstream KH("eigvec.txt");
  if (!KH.is_open()) {
    std::cout << "Error opening file " << "eigvec.txt" << std::endl;
    throw std::runtime_error("");
  }

  int nKH = 128 + 2;
  KH >> nKH;
  nKH += 2;
  real kxKH = 1.0;
  KH >> kxKH;

  Array<real> *yKH = new Array<real>(1, 0, nKH);
  Array<real> *densReal = new Array<real>(1, 0, nKH);
  Array<real> *densImag = new Array<real>(1, 0, nKH);
  Array<real> *velxReal = new Array<real>(1, 0, nKH);
  Array<real> *velxImag = new Array<real>(1, 0, nKH);
  Array<real> *velyReal = new Array<real>(1, 0, nKH);
  Array<real> *velyImag = new Array<real>(1, 0, nKH);

  real *pyKH = yKH->GetPointer();
  real *pdR  = densReal->GetPointer();
  real *pdI  = densImag->GetPointer();
  real *puR  = velxReal->GetPointer();
  real *puI  = velxImag->GetPointer();
  real *pvR  = velyReal->GetPointer();
  real *pvI  = velyImag->GetPointer();

  for (int j = 1; j < nKH - 1; j++)
    KH >> pyKH[j] >> pdR[j] >> pdI[j] >> puR[j] >> puI[j] >> pvR[j] >> pvI[j];

  KH.close();

  pyKH[0] = pyKH[1] - (pyKH[2] - pyKH[1]);
  pdR[0] = pdR[nKH - 2];
  pdI[0] = pdI[nKH - 2];
  puR[0] = puR[nKH - 2];
  puI[0] = puI[nKH - 2];
  pvR[0] = pvR[nKH - 2];
  pvI[0] = pvI[nKH - 2];

  pyKH[nKH - 1] = pyKH[nKH - 2] + (pyKH[2] - pyKH[1]);
  pdR[nKH - 1] = pdR[1];
  pdI[nKH - 1] = pdI[1];
  puR[nKH - 1] = puR[1];
  puI[nKH - 1] = puI[1];
  pvR[nKH - 1] = pvR[1];
  pvI[nKH - 1] = pvI[1];

  real miny = mesh->GetMinY();
  real maxy = mesh->GetMaxY();

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devAddEigenVector<realNeq, CL>,
                                       (size_t) 0, 0);

    devAddEigenVector<realNeq, CL><<<nBlocks, nThreads>>>
      (nVertex, pVc, pState, pdR, pdI, puR, puI, pvR, pvI,
       pyKH[1] - pyKH[0], kxKH, pyKH, miny, maxy, G, G - 1.0);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (unsigned int n = 0; n < nVertex; n++)
      AddEigenVectorSingle(n, pVc, pState, pdR, pdI, puR, puI, pvR, pvI,
                           pyKH[1] - pyKH[0], kxKH, pyKH, miny, maxy,
                           G, G - 1.0);
  }

  delete yKH;
  delete densReal;
  delete densImag;
  delete velxReal;
  delete velxImag;
  delete velyReal;
  delete velyImag;
}

//##############################################################################
// Instantiate
//##############################################################################

template void Simulation<real, CL_ADVECT>::KHAddEigenVector();
template void Simulation<real, CL_BURGERS>::KHAddEigenVector();
template void Simulation<real3, CL_CART_ISO>::KHAddEigenVector();
template void Simulation<real4, CL_CART_EULER>::KHAddEigenVector();

}  // namespace astrix
