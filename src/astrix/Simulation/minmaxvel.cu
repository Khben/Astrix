#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file minmaxvel.cu
\brief File containing functions to find minimum and maximum velocity in the Simulation

*/ /* \section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "../Mesh/mesh.h"
#include "./simulation.h"
#include "../Common/cudaLow.h"

namespace astrix {

//#########################################################################
/*! \brief Find minimum/maximum velocity at vertex \a i

\param i Vertex to consider
\param *pState Pointer to vertex state
\param *pMinVel Pointer to minimum velocities (output)
\param *pMaxVel Pointer to maximum velocities (output)*/
//#########################################################################

template<ConservationLaw CL>
__host__ __device__
void FillMinMaxVelocitySingle(unsigned int i, real4 *pState,
                              real *pMinVel, real *pMaxVel)
{
  real dens = pState[i].x;
  real momx = pState[i].y;
  real momy = pState[i].z;

  // Assume maximum is x-velocity
  real vMax = momx/dens;
  real vMin = momy/dens;

  // Swap if necessary
  if (vMin > vMax) {
    real vTemp = vMax;
    vMax = vMin;
    vMin = vTemp;
  }

  // Output
  pMinVel[i] = vMin;
  pMaxVel[i] = vMax;
}

//! Version for three equations
template<ConservationLaw CL>
__host__ __device__
void FillMinMaxVelocitySingle(unsigned int i, real3 *pState,
                              real *pMinVel, real *pMaxVel)
{
  real dens = pState[i].x;
  real momx = pState[i].y;
  real momy = pState[i].z;

  // Assume maximum is x-velocity
  real vMax = momx/dens;
  real vMin = momy/dens;

  // Swap if necessary
  if (vMin > vMax) {
    real vTemp = vMax;
    vMax = vMin;
    vMin = vTemp;
  }

  // Output
  pMinVel[i] = vMin;
  pMaxVel[i] = vMax;
}

//! Version for single equation
template<ConservationLaw CL>
__host__ __device__
void FillMinMaxVelocitySingle(unsigned int i, real *pState,
                              real *pMinVel, real *pMaxVel)
{
  pMinVel[i] = (real) 1.0;
  pMaxVel[i] = (real) 1.0;

  if (CL == CL_BURGERS) {
    pMinVel[i] = pState[i];
    pMaxVel[i] = pState[i];
  }
}

//#########################################################################
/*! \brief Fill minimum/maximum velocity arrays

\param nVertex Total number of vertices in Mesh
\param *pState Pointer to vertex state
\param *pMinVel Pointer to minimum velocities (output)
\param *pMaxVel Pointer to maximum velocities (output)*/
//#########################################################################

template<class realNeq, ConservationLaw CL>
__global__ void
devFillMinMaxVelocity(unsigned int nVertex, realNeq *pState,
                      real *pMinVel, real *pMaxVel)
{
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nVertex) {
    FillMinMaxVelocitySingle<CL>(i, pState, pMinVel, pMaxVel);

    i += gridDim.x*blockDim.x;
  }
}

//#########################################################################
/*! Returns minimum and maximum velocity on Mesh*/
//#########################################################################

template <class realNeq, ConservationLaw CL>
real2 Simulation<realNeq, CL>::FindMinMaxVelocity()
{
  unsigned int nVertex = mesh->GetNVertex();

  // State at vertices
  realNeq *pState = vertexState->GetPointer();

  // Arrays containing minimum/maximum velocity for each vertex
  Array<real> *minVelocity = new Array<real>(1, cudaFlag, nVertex);
  Array<real> *maxVelocity = new Array<real>(1, cudaFlag, nVertex);

  real *pMinVel = minVelocity->GetPointer();
  real *pMaxVel = maxVelocity->GetPointer();

  // Fill minimum/maximum velocity arrays
  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devFillMinMaxVelocity<realNeq, CL>,
                                       (size_t) 0, 0);

    devFillMinMaxVelocity<realNeq, CL><<<nBlocks, nThreads>>>
      (nVertex, pState, pMinVel, pMaxVel);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (unsigned int i = 0; i < nVertex; i++)
      FillMinMaxVelocitySingle<CL>(i, pState, pMinVel, pMaxVel);
  }

  // Find minimum/maximum velocity
  real2 minmaxvel;
  minmaxvel.x = minVelocity->Minimum();
  minmaxvel.y = maxVelocity->Maximum();

  delete minVelocity;
  delete maxVelocity;

  return minmaxvel;
}

//##############################################################################
// Instantiate
//##############################################################################

template real2 Simulation<real, CL_ADVECT>::FindMinMaxVelocity();
template real2 Simulation<real, CL_BURGERS>::FindMinMaxVelocity();
template real2 Simulation<real3, CL_CART_ISO>::FindMinMaxVelocity();
template real2 Simulation<real3, CL_CYL_ISO>::FindMinMaxVelocity();
template real2 Simulation<real4, CL_CART_EULER>::FindMinMaxVelocity();

}  // namespace astrix
