#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file minmaxvel.cu
\brief File containing functions to find minimum and maximum velocity in the Simulation
*/

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "../Mesh/mesh.h"
#include "./simulation.h"
#include "../Common/cudaLow.h"

namespace astrix {

//#########################################################################
/*! \brief Find minimum/maximum velocity at vertex \a i

\param i Vertex to consider
\param *pState Pointer to vertex state
\param *pMinVel Pointer to minimum velocities (output)
\param *pMinVel Pointer to maximum velocities (output)*/
//#########################################################################

__host__ __device__
void FillMinMaxVelocitySingle(unsigned int i, real4 *pState, 
			      real *pMinVel, real *pMaxVel)
{
  real dens = pState[i].x;
  real momx = pState[i].y;
  real momy = pState[i].z;

  // Assume maximum is x-velocity
  real vMax = momx/dens;
  real vMin = momy/dens;

  // Swap if necessary
  if (vMin > vMax) {
    real vTemp = vMax;
    vMax = vMin;
    vMin = vTemp;
  }

  // Output
  pMinVel[i] = vMin;
  pMaxVel[i] = vMax;
}

//#########################################################################
/*! \brief Fill minimum/maximum velocity arrays

\param nVertex Total number of vertices in Mesh
\param *pState Pointer to vertex state
\param *pMinVel Pointer to minimum velocities (output)
\param *pMinVel Pointer to maximum velocities (output)*/
//#########################################################################

__global__ void 
devFillMinMaxVelocity(unsigned int nVertex, real4 *pState, 
		      real *pMinVel, real *pMaxVel)
{
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nVertex) {
    FillMinMaxVelocitySingle(i, pState, pMinVel, pMaxVel);
    
    i += gridDim.x*blockDim.x;
  }
}

//#########################################################################
/*! \brief Find minimum and maximum velocity on Mesh

\param minVel Will contain minimum velocity (output)
\param maxVel Will contain maximum velocity (output)*/
//#########################################################################

void Simulation::FindMinMaxVelocity(real& minVel, real& maxVel)
{
  unsigned int nVertex = mesh->GetNVertex();
  
  // State at vertices
  real4 *pState = vertexState->GetPointer(); 

  // Arrays containing minimum/maximum velocity for each vertex
  Array<real> *minVelocity = new Array<real>(1, cudaFlag, nVertex);
  Array<real> *maxVelocity = new Array<real>(1, cudaFlag, nVertex);
  
  real *pMinVel = minVelocity->GetPointer();
  real *pMaxVel = maxVelocity->GetPointer();

  // Fill minimum/maximum velocity arrays
  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
				       devFillMinMaxVelocity, 
				       (size_t) 0, 0);

    devFillMinMaxVelocity<<<nBlocks, nThreads>>>
      (nVertex, pState, pMinVel, pMaxVel);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (unsigned int i = 0; i < nVertex; i++) 
      FillMinMaxVelocitySingle(i, pState, pMinVel, pMaxVel);
  }

  // Find minimum/maximum velocity
  minVel = minVelocity->Minimum(); 
  maxVel = maxVelocity->Maximum();

  delete minVelocity;
  delete maxVelocity;
}

}
