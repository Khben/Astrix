#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file mass.cu
\brief Functions to calculate total mass in Simulation

\section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/
#include <iostream>

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "../Mesh/mesh.h"
#include "./simulation.h"
#include "../Common/cudaLow.h"

namespace astrix {

//######################################################################
/*! \brief Compute mass associated with vertex \a n

\param n Vertex to consider
\param *pState Pointer to state vector
\param *pVarea Pointer to vertex area
\param *pVm Pointer to vertex mass (output)*/
//######################################################################

__host__ __device__
void FillMassArraySingle(unsigned int n, real4 *pState,
                         const real *pVarea, real *pVm)
{
  pVm[n] = pVarea[n]*pState[n].x;
}

__host__ __device__
void FillMassArraySingle(unsigned int n, real *pState,
                         const real *pVarea, real *pVm)
{
  pVm[n] = pVarea[n]*pState[n];
}

//######################################################################
/*! \brief Compute mass associated with vertices

\param nVertex Total number of vertices in Mesh
\param *pState Pointer to state vector
\param *pVarea Pointer to vertex area
\param *pVm Pointer to vertex mass (output)*/
//######################################################################

__global__ void
devFillMassArray(unsigned int nVertex, realNeq *pState,
                 const real *pVarea, real *pVm)
{
  // n=vertex number
  unsigned int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nVertex) {
    FillMassArraySingle(n, pState, pVarea, pVm);
    n += blockDim.x*gridDim.x;
  }
}

//######################################################################
/*! \brief Compute total mass in simulation*/
//######################################################################

real Simulation::TotalMass()
{
  unsigned int nVertex = mesh->GetNVertex();
  realNeq *pState = vertexState->GetPointer();

  // Mass in every cell
  Array<real> *vertexMass = new Array<real>(1, cudaFlag, nVertex);
  real *pVm = vertexMass->GetPointer();

  const real *pVarea = mesh->VertexAreaData();

  if (cudaFlag == 1) {
    int nThreads = 128;
    int nBlocks  = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devFillMassArray,
                                       (size_t) 0, 0);

    // Execute kernel...
    devFillMassArray<<<nBlocks, nThreads>>>
      (nVertex, pState, pVarea, pVm);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
  } else {
    for (unsigned int n = 0; n < nVertex; n++)
      FillMassArraySingle(n, pState, pVarea, pVm);
  }

  real mass = vertexMass->Sum();

  delete vertexMass;

  return mass;
}

}  // namespace astrix
