#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file selectlump.cu

\section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/
#include <iostream>

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "../Mesh/mesh.h"
#include "./simulation.h"
#include "../Common/cudaLow.h"
#include "../Common/inlineMath.h"
#include "../Common/hip/hip_vector_types.h"

namespace astrix {

//######################################################################
/*! \brief Computing contribution of second mass matrix and selective lumping for single triangle

\param n Triangle to consider
\param dt Time step
\param massMatrix Mass matrix used
\param selectLumpFlag Flag whether to use selective lumping
\param *pTv Pointer to triangle vertices
\param *pDstate Pointer to state difference at vertices
\param *pTresLDA0 Triangle residue LDA direction 0
\param *pTresLDA1 Triangle residue LDA direction 1
\param *pTresLDA2 Triangle residue LDA direction 2
\param *pTresN0 Triangle residue N direction 0
\param *pTresN1 Triangle residue N direction 1
\param *pTresN2 Triangle residue N direction 2
\param *pTl Pointer to triangle edge lengths
\param nVertex Total number of vertices in Mesh*/
//######################################################################

template<class realNeq, ConservationLaw CL>
__host__ __device__
void SelectLumpSingle(int n, real dt, int massMatrix, int selectLumpFlag,
                      const int3* __restrict__ pTv, realNeq *pDstate,
                      realNeq *pTresLDA0, realNeq *pTresLDA1,
                      realNeq *pTresLDA2, realNeq *pTresN0,
                      realNeq *pTresN1, realNeq *pTresN2,
                      const real3 *pTl, int nVertex)
{
  real half = (real) 0.5;
  real two = (real) 2.0;

  real f = (real) 0.0;
  if (selectLumpFlag == 1) f += (real) 1.0/(real) 12.0;
  if (massMatrix == 2) f += (real) 1.0/ (real) 36.0;

  // Vertices belonging to triangle: 3 coalesced reads
  int v1 = pTv[n].x;
  int v2 = pTv[n].y;
  int v3 = pTv[n].z;
  while (v1 >= nVertex) v1 -= nVertex;
  while (v2 >= nVertex) v2 -= nVertex;
  while (v3 >= nVertex) v3 -= nVertex;
  while (v1 < 0) v1 += nVertex;
  while (v2 < 0) v2 += nVertex;
  while (v3 < 0) v3 += nVertex;

  realNeq dW0 = pDstate[v1];
  realNeq dW1 = pDstate[v2];
  realNeq dW2 = pDstate[v3];

  real Tl1 = pTl[n].x;
  real Tl2 = pTl[n].y;
  real Tl3 = pTl[n].z;

  // Calculate triangle area
  real s = half*(Tl1 + Tl2 + Tl3);
  // |T|/(12*dt)
  real Adt = sqrt(s*(s - Tl1)*(s - Tl2)*(s - Tl3))*f/dt;

  realNeq ResLDA0 = (-two*dW0 + dW1 + dW2)*Adt;
  realNeq ResLDA1 = (dW0 - two*dW1 + dW2)*Adt;
  realNeq ResLDA2 = (dW0 + dW1 - two*dW2)*Adt;

  pTresLDA0[n] -= ResLDA0/Tl1;
  pTresLDA1[n] -= ResLDA1/Tl2;
  pTresLDA2[n] -= ResLDA2/Tl3;

  if (selectLumpFlag == 1) {
    pTresN0[n] -= ResLDA0/Tl1;
    pTresN1[n] -= ResLDA1/Tl2;
    pTresN2[n] -= ResLDA2/Tl3;
  }
}

//######################################################################
/*! \brief Kernel computing contribution of second mass matrix and selective lumping for all triangles

\param nTriangle Total number of triangles in Mesh
\param dt Time step
\param massMatrix Mass matrix used
\param selectLumpFlag Flag whether to use selective lumping
\param *pTv Pointer to triangle vertices
\param *pDstate Pointer to state difference at vertices
\param *pTresLDA0 Triangle residue LDA direction 0
\param *pTresLDA1 Triangle residue LDA direction 1
\param *pTresLDA2 Triangle residue LDA direction 2
\param *pTresN0 Triangle residue N direction 0
\param *pTresN1 Triangle residue N direction 1
\param *pTresN2 Triangle residue N direction 2
\param *pTl Pointer to triangle edge lengths
\param nVertex Total number of vertices in Mesh*/
//######################################################################

template<class realNeq, ConservationLaw CL>
__global__ void
devSelectLump(int nTriangle, real dt, int massMatrix, int selectLumpFlag,
              const int3* __restrict__ pTv, realNeq *pDstate,
              realNeq *pTresLDA0, realNeq *pTresLDA1, realNeq *pTresLDA2,
              realNeq *pTresN0, realNeq *pTresN1, realNeq *pTresN2,
              const real3 *pTl, int nVertex)
{
  int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nTriangle) {
    SelectLumpSingle<realNeq, CL>(n, dt, massMatrix, selectLumpFlag,
                                  pTv, pDstate,
                                  pTresLDA0, pTresLDA1, pTresLDA2,
                                  pTresN0, pTresN1, pTresN2,
                                  pTl, nVertex);

    // Next triangle
    n += blockDim.x*gridDim.x;
  }
}

//######################################################################
/*! Add contribution of selective lumping and the second mass matrix to residuals. If \a massMatrix is not equal to 2 and \a selectLumpFlag is not equal to 1 nothing happens.

\param dt Time step
\param massMatrix Mass matrix used
\param selectLumpFlag Flag whether to use selective lumping*/
//######################################################################

template <class realNeq, ConservationLaw CL>
void Simulation<realNeq, CL>::SelectLump(real dt, int massMatrix,
                                         int selectLumpFlag)
{
  int nTriangle = mesh->GetNTriangle();
  int nVertex = mesh->GetNVertex();

  realNeq *pDstate = vertexStateDiff->GetPointer();
  realNeq *pTresLDA0 = triangleResidueLDA->GetPointer(0);
  realNeq *pTresLDA1 = triangleResidueLDA->GetPointer(1);
  realNeq *pTresLDA2 = triangleResidueLDA->GetPointer(2);
  realNeq *pTresN0 = triangleResidueN->GetPointer(0);
  realNeq *pTresN1 = triangleResidueN->GetPointer(1);
  realNeq *pTresN2 = triangleResidueN->GetPointer(2);

  const int3 *pTv = mesh->TriangleVerticesData();
  const real3 *pTl  = mesh->TriangleEdgeLengthData();

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devSelectLump<realNeq, CL>,
                                       (size_t) 0, 0);

    devSelectLump<realNeq, CL><<<nBlocks, nThreads>>>
      (nTriangle, dt, massMatrix, selectLumpFlag,
       pTv, pDstate,
       pTresLDA0, pTresLDA1, pTresLDA2,
       pTresN0, pTresN1, pTresN2,
       pTl, nVertex);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int n = 0; n < nTriangle; n++)
      SelectLumpSingle<realNeq, CL>(n, dt, massMatrix, selectLumpFlag,
                                    pTv, pDstate,
                                    pTresLDA0, pTresLDA1, pTresLDA2,
                                    pTresN0, pTresN1, pTresN2,
                                    pTl, nVertex);
  }
}

//##############################################################################
// Instantiate
//##############################################################################

template void Simulation<real, CL_ADVECT>::SelectLump(real dt,
                                                      int massMatrix,
                                                      int selectLumpFlag);
template void Simulation<real, CL_BURGERS>::SelectLump(real dt,
                                                       int massMatrix,
                                                       int selectLumpFlag);
template void Simulation<real3, CL_CART_ISO>::SelectLump(real dt,
                                                         int massMatrix,
                                                         int selectLumpFlag);
template void Simulation<real4, CL_CART_EULER>::SelectLump(real dt,
                                                           int massMatrix,
                                                           int selectLumpFlag);

}  // namespace astrix
