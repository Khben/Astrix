#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file unphysical.cu
\brief File containing functions detect unphysical states at vertices.*/
#include <iostream>

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "../Mesh/mesh.h"
#include "./simulation.h"
#include "../Common/cudaLow.h"
#include "../Common/inlineMath.h"

namespace astrix {

//######################################################################
/*! \brief Check unphysical state at vertex \a v

\param v Vertex to consider
\param *pState Pointer to state at vertices
\param *pVertexUnphysicalFlag Pointer to array of flags indicating whether state is physical (0) or unphysical (1) (output)
\param G1 Ratio of specific heats - 1*/
//######################################################################

__host__ __device__
void FlagUnphysicalVertex(const int v, real4 *pState,
			  int *pVertexUnphysicalFlag, const real G1)
{
  const real zero = (real) 0.0;
  const real half = (real) 0.5;

  // Assume everything is fine
  int ret = 0;

  real dens = pState[v].x;
  real momx = pState[v].y;
  real momy = pState[v].z;
  real ener = pState[v].w;

  // Pressure
  real p = G1*(ener - half*(Sq(momx) + Sq(momy))/dens);
  
  // Flag if negative density or pressure
  if(dens < zero || p < zero || isnan(p)) ret = 1;
  
  // Output flag
  pVertexUnphysicalFlag[v] = ret;
}

__host__ __device__
void FlagUnphysicalVertex(const int v, real *pState,
			  int *pVertexUnphysicalFlag, const real G1)
{
  // Output flag
  pVertexUnphysicalFlag[v] = 0;
}

//######################################################################
/*! \brief Kernel checking vertices for unphysical state

\param nVertex Total number of vertices in Mesh
\param *pState Pointer to state at vertices
\param *pVertexUnphysicalFlag Pointer to array of flags indicating whether state is physical (0) or unphysical (1) (output)
\param G1 Ratio of specific heats - 1*/ 
//######################################################################

__global__ void 
devFlagUnphysical(const int nVertex, realNeq *pState,
		  int *pVertexUnphysicalFlag, const real G1)
{
  // n=vertex number
  int n = blockIdx.x*blockDim.x + threadIdx.x; 

  while (n < nVertex) {
    FlagUnphysicalVertex(n, pState, pVertexUnphysicalFlag, G1);

    n += blockDim.x*gridDim.x;
  }
}

//######################################################################
/*! Check all vertices for unphysical state. 

  \param *pVertexUnphysicalFlag Pointer to array of flags indicating whether state is physical (0) or unphysical (1) (output)*/
//######################################################################

void Simulation::FlagUnphysical(Array<int> *vertexUnphysicalFlag)
{
  // Total number of vertices in Mesh
  int nVertex = mesh->GetNVertex();

  // State vector at vertices
  realNeq *state = vertexState->GetPointer();

  // Pointer to output
  int *pVertexUnphysicalFlag = vertexUnphysicalFlag->GetPointer();

  if (cudaFlag == 1) {
    int nThreads = 128;
    int nBlocks  = 128;
    
    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
				       devFlagUnphysical, 
				       (size_t) 0, 0);
    
    // Execute kernel... 
    devFlagUnphysical<<<nBlocks,nThreads>>>
      (nVertex, state, pVertexUnphysicalFlag, specificHeatRatio - 1.0);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int v = 0; v < nVertex; v++)
      FlagUnphysicalVertex(v, state, pVertexUnphysicalFlag,
			   specificHeatRatio - 1.0);
  }
}

}
