#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file unphysical.cu
\brief File containing functions detect unphysical states at vertices.

*/ /* \section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/
#include <iostream>

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "../Mesh/mesh.h"
#include "./simulation.h"
#include "../Common/cudaLow.h"
#include "../Common/inlineMath.h"
#include "./Param/simulationparameter.h"

namespace astrix {

//######################################################################
/*! \brief Check unphysical state at vertex \a v

\param v Vertex to consider
\param *pState Pointer to state at vertices
\param *pVp Pointer to external potential at vertices
\param *pVertexUnphysicalFlag Pointer to array of flags indicating whether state is physical (0) or unphysical (1) (output)
\param G1 Ratio of specific heats - 1*/
//######################################################################

__host__ __device__
void FlagUnphysicalVertex(const int v, real4 *pState, real *pVp,
                          int *pVertexUnphysicalFlag, const real G1)
{
  const real zero = (real) 0.0;
  const real half = (real) 0.5;

  // Assume everything is fine
  int ret = 0;

  real dens = pState[v].x;
  real momx = pState[v].y;
  real momy = pState[v].z;
  real ener = pState[v].w;

  // Pressure
  real p = G1*(ener - half*(Sq(momx) + Sq(momy))/dens - dens*pVp[v]);

  // Flag if negative density or pressure
  if (dens < zero || p < zero || isnan(p)) ret = 1;

  // Output flag
  pVertexUnphysicalFlag[v] = ret;
}

__host__ __device__
void FlagUnphysicalVertex(const int v, real3 *pState, real *pVp,
                          int *pVertexUnphysicalFlag, const real G1)
{
  const real zero = (real) 0.0;

  // Assume everything is fine
  int ret = 0;

  real dens = pState[v].x;

  // Flag if negative density or pressure
  if (dens < zero) ret = 1;

  // Output flag
  pVertexUnphysicalFlag[v] = ret;
}

__host__ __device__
void FlagUnphysicalVertex(const int v, real *pState, real *pVp,
                          int *pVertexUnphysicalFlag, const real G1)
{
  // Output flag
  pVertexUnphysicalFlag[v] = 0;
}

//######################################################################
/*! \brief Kernel checking vertices for unphysical state

\param nVertex Total number of vertices in Mesh
\param *pState Pointer to state at vertices
\param *pVp Pointer to external potential at vertices
\param *pVertexUnphysicalFlag Pointer to array of flags indicating whether state is physical (0) or unphysical (1) (output)
\param G1 Ratio of specific heats - 1*/
//######################################################################

template<class realNeq, ConservationLaw CL>
__global__ void
devFlagUnphysical(const int nVertex, realNeq *pState, real *pVp,
                  int *pVertexUnphysicalFlag, const real G1)
{
  // n=vertex number
  int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nVertex) {
    FlagUnphysicalVertex(n, pState, pVp, pVertexUnphysicalFlag, G1);

    n += blockDim.x*gridDim.x;
  }
}

//######################################################################
/*! Check all vertices for unphysical state.

  \param *vertexUnphysicalFlag Pointer to Array of flags indicating whether state is physical (0) or unphysical (1) (output)*/
//######################################################################

template <class realNeq, ConservationLaw CL>
void Simulation<realNeq, CL>::FlagUnphysical(Array<int> *vertexUnphysicalFlag)
{
  // Total number of vertices in Mesh
  int nVertex = mesh->GetNVertex();

  // State vector at vertices
  realNeq *state = vertexState->GetPointer();
  real *pVp = vertexPotential->GetPointer();

  // Ratio of specific heats
  real G = simulationParameter->specificHeatRatio;

  // Pointer to output
  int *pVertexUnphysicalFlag = vertexUnphysicalFlag->GetPointer();

  if (cudaFlag == 1) {
    int nThreads = 128;
    int nBlocks  = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devFlagUnphysical<realNeq, CL>,
                                       (size_t) 0, 0);

    // Execute kernel...
    devFlagUnphysical<realNeq, CL><<<nBlocks, nThreads>>>
      (nVertex, state, pVp, pVertexUnphysicalFlag, G - 1.0);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int v = 0; v < nVertex; v++)
      FlagUnphysicalVertex(v, state, pVp, pVertexUnphysicalFlag, G - 1.0);
  }
}

//##############################################################################
// Instantiate
//##############################################################################

template void Simulation<real, CL_ADVECT>::
FlagUnphysical(Array<int> *vertexUnphysicalFlag);
template void Simulation<real, CL_BURGERS>::
FlagUnphysical(Array<int> *vertexUnphysicalFlag);
template void Simulation<real3, CL_CART_ISO>::
FlagUnphysical(Array<int> *vertexUnphysicalFlag);
template void Simulation<real3, CL_CYL_ISO>::
FlagUnphysical(Array<int> *vertexUnphysicalFlag);
template void Simulation<real4, CL_CART_EULER>::
FlagUnphysical(Array<int> *vertexUnphysicalFlag);

}  // namespace astrix
