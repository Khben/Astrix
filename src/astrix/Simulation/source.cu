#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file source.cu
\brief File containing function to calculate source term contribution to residual.

*/ /* \section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/

#include <iostream>

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "../Mesh/mesh.h"
#include "./simulation.h"
#include "../Common/cudaLow.h"
#include "../Common/inlineMath.h"
#include "../Common/hip/hip_vector_types.h"
#include "./Param/simulationparameter.h"

namespace astrix {

//######################################################################
/*! \brief Calculating source term at vertex n.

\param n Vertex to consider
\param problemDef Problem definition
\param *pVc Pointer tovertex coordinates
\param *pVp Pointer to external potential at vertices
\param *pState Pointer to state vector
\param *pSource Pointer to source vector (output)*/
//######################################################################

template<ConservationLaw CL>
__host__ __device__
void CalcVertexSourceSingle(int n, ProblemDefinition problemDef,
                            const real2 *pVc, const real *pVp,
                            const real4 *pState, real4 *pSource)
{
  pSource[n].x = 0.0;
  pSource[n].y = 0.0;
  pSource[n].z = 0.0;
  pSource[n].w = 0.0;

  if (problemDef == PROBLEM_DISC) {
    real x = pVc[n].x;
    real y = pVc[n].y;

    real r = exp(x);
    pSource[n].y =
      (Sq(pState[n].z)/Sq(Sq(r)) - Sq(pState[n].y))/pState[n].x -
      pState[n].x/Cb(r);
  }

}

//! Version for three equations
template<ConservationLaw CL>
__host__ __device__
void CalcVertexSourceSingle(int n, ProblemDefinition problemDef, const real2 *pVc,
                            const real *pVp, const real3 *pState, real3 *pSource)
{
  pSource[n].x = 0.0;
  pSource[n].y = 0.0;
  pSource[n].z = 0.0;

  if (problemDef == PROBLEM_DISC) {
    real x = pVc[n].x;
    real y = pVc[n].y;

    // Cylindrical radius
    real r = exp(x);
    // Centrifugal, gravity, geometrical
    pSource[n].y =
      (0.0*Sq(pState[n].z)/Sq(Sq(r)) - Sq(pState[n].y))/pState[n].x -
      0.0*pState[n].x/Cb(r);
  }

}

//! Version for single equation
template<ConservationLaw CL>
__host__ __device__
void CalcVertexSourceSingle(int n, ProblemDefinition problemDef, const real2 *pVc,
                            const real *pVp, const real *pState, real *pSource)
{
  pSource[n] = 0.0;

  if (problemDef == PROBLEM_SOURCE) {
    if (CL == CL_ADVECT) {
      pSource[n] = Sq(pState[n]);
      //pSource[n] = 8.0*M_PI*cos(8.0*M_PI*pVc[n].x);

      //real N = 10.0;
      //real x = pVc[n].x;
      //pSource[n] += 0.0*(N*cos(n*x)*(1.0 + x) - sin(N*x))/Sq(1.0 + x) -
      //  0.0*sin(N*x)/(1.0 + x);
    }

    if (CL == CL_BURGERS)
      pSource[n] = Sq(pState[n]);

  }
}

//######################################################################
/*! \brief Kernel calculating source term at vertices.

\param nVertex Total number of vertices in Mesh
\param problemDef Problem definition
\param *pVc Pointer to vertex coordinates
\param *pVp Pointer to external potential at vertices
\param *pState Pointer to state vector
\param *pSource Pointer to source vector (output)*/
//######################################################################

template<class realNeq, ConservationLaw CL>
__global__ void
devCalcVertexSource(int nVertex, ProblemDefinition problemDef, const real2 *pVc,
                    const real *pVp, const realNeq *pState, realNeq *pSource)
{
  // n = vertex number
  int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nVertex) {
    CalcVertexSourceSingle<CL>(n, problemDef, pVc, pVp, pState, pSource);

    n += blockDim.x*gridDim.x;
  }
}

//######################################################################
/*! \brief Calculating source term contribution to residual for single triangle.

\param n Triangle to consider
\param problemDef Problem definition
\param nVertex Total number of vertices in Mesh
\param *pTv Pointer to triangle vertices
\param *pTn1 Pointer first triangle edge normal
\param *pTn2 Pointer second triangle edge normal
\param *pTn3 Pointer third triangle edge normal
\param *pTl Pointer to triangle edge lengths
\param *pVp Pointer to external potential at vertices
\param *pState Pointer to state vector
\param *pSource Pointer to source vector (output)
\param *pVs Pointer to vertex source term*/
//######################################################################

template<ConservationLaw CL>
__host__ __device__
void CalcSourceSingle(int n, ProblemDefinition problemDef,
                      int nVertex, const int3 *pTv,
                      const real2 *pTn1, const real2 *pTn2, const real2 *pTn3,
                      const real3 *pTl, const real *pVp,
                      const real4 *pState, real4 *pSource, const real4 *pVs)
{
  pSource[n].x = 0.0;
  pSource[n].y = 0.0;
  pSource[n].z = 0.0;
  pSource[n].w = 0.0;

  if (problemDef == PROBLEM_SOURCE) {
    real three = (real) 3.0;
    real half = (real) 0.5;

    // Vertices belonging to triangle
    int v1 = pTv[n].x;
    int v2 = pTv[n].y;
    int v3 = pTv[n].z;
    while (v1 >= nVertex) v1 -= nVertex;
    while (v2 >= nVertex) v2 -= nVertex;
    while (v3 >= nVertex) v3 -= nVertex;
    while (v1 < 0) v1 += nVertex;
    while (v2 < 0) v2 += nVertex;
    while (v3 < 0) v3 += nVertex;

    real tl1 = pTl[n].x;
    real tl2 = pTl[n].y;
    real tl3 = pTl[n].z;

    real d1 = pState[v1].x;
    real d2 = pState[v2].x;
    real d3 = pState[v3].x;
    real dG = (d1 + d2 + d3)/three;

    /*
    real m1 = pState[v1].y;
    real m2 = pState[v2].y;
    real m3 = pState[v3].y;
    real mG = (m1 + m2 + m3)/three;

    real n1 = pState[v1].z;
    real n2 = pState[v2].z;
    real n3 = pState[v3].z;
    real nG = (n1 + n2 + n3)/three;
    */

    real dpotdx = half*
      (pVp[v1]*pTn1[n].x*tl1 +
       pVp[v2]*pTn2[n].x*tl2 +
       pVp[v3]*pTn3[n].x*tl3);
    real dpotdy = half*
      (pVp[v1]*pTn1[n].y*tl1 +
       pVp[v2]*pTn2[n].y*tl2 +
       pVp[v3]*pTn3[n].y*tl3);

    pSource[n].x = 0.0;
    pSource[n].y = dG*dpotdx;
    pSource[n].z = dG*dpotdy;
    pSource[n].w = 0.0;//mG*dpotdx + nG*dpotdy;
  }

}

//! Version for three equations
template<ConservationLaw CL>
__host__ __device__
void CalcSourceSingle(int n, ProblemDefinition problemDef,
                      int nVertex, const int3 *pTv,
                      const real2 *pTn1, const real2 *pTn2, const real2 *pTn3,
                      const real3 *pTl, const real *pVp,
                      const real3 *pState, real3 *pSource, const real3 *pVs)
{
  pSource[n].x = 0.0;
  pSource[n].y = 0.0;
  pSource[n].z = 0.0;

  if (problemDef == PROBLEM_SOURCE) {
    real three = (real) 3.0;
    real half = (real) 0.5;

    // Vertices belonging to triangle
    int v1 = pTv[n].x;
    int v2 = pTv[n].y;
    int v3 = pTv[n].z;
    while (v1 >= nVertex) v1 -= nVertex;
    while (v2 >= nVertex) v2 -= nVertex;
    while (v3 >= nVertex) v3 -= nVertex;
    while (v1 < 0) v1 += nVertex;
    while (v2 < 0) v2 += nVertex;
    while (v3 < 0) v3 += nVertex;

    real tl1 = pTl[n].x;
    real tl2 = pTl[n].y;
    real tl3 = pTl[n].z;

    real d1 = pState[v1].x;
    real d2 = pState[v2].x;
    real d3 = pState[v3].x;
    real dG = (d1 + d2 + d3)/three;

    real dpotdx = half*
      (pVp[v1]*pTn1[n].x*tl1 +
       pVp[v2]*pTn2[n].x*tl2 +
       pVp[v3]*pTn3[n].x*tl3);
    real dpotdy = half*
      (pVp[v1]*pTn1[n].y*tl1 +
       pVp[v2]*pTn2[n].y*tl2 +
       pVp[v3]*pTn3[n].y*tl3);

    pSource[n].x = 0.0;
    pSource[n].y = dG*dpotdx;
    pSource[n].z = dG*dpotdy;
  }

  if (problemDef == PROBLEM_DISC) {
    real three = (real) 3.0;
    real half = (real) 0.5;

    // Vertices belonging to triangle
    int v1 = pTv[n].x;
    int v2 = pTv[n].y;
    int v3 = pTv[n].z;
    while (v1 >= nVertex) v1 -= nVertex;
    while (v2 >= nVertex) v2 -= nVertex;
    while (v3 >= nVertex) v3 -= nVertex;
    while (v1 < 0) v1 += nVertex;
    while (v2 < 0) v2 += nVertex;
    while (v3 < 0) v3 += nVertex;

    real tl1 = pTl[n].x;
    real tl2 = pTl[n].y;
    real tl3 = pTl[n].z;

    // Average source term
    real3 S = (pVs[v1] + pVs[v2] + pVs[v3])/three;

    // Triangle area
    real s = half*(tl1 + tl2 + tl3);
    real area = sqrt(s*(s - tl1)*(s - tl2)*(s - tl3));

    // Source contribution to residual (note minus sign!)
    pSource[n] = -S*area;
  }

}

//! Version for single equation
template<ConservationLaw CL>
__host__ __device__
void CalcSourceSingle(int n, ProblemDefinition problemDef,
                      int nVertex, const int3 *pTv,
                      const real2 *pTn1, const real2 *pTn2, const real2 *pTn3,
                      const real3 *pTl, const real *pVp,
                      const real *pState, real *pSource, const real *pVs)
{
  pSource[n] = 0.0;

  if (problemDef == PROBLEM_SOURCE) {
    real three = (real) 3.0;

    // Vertices belonging to triangle
    int v1 = pTv[n].x;
    int v2 = pTv[n].y;
    int v3 = pTv[n].z;
    while (v1 >= nVertex) v1 -= nVertex;
    while (v2 >= nVertex) v2 -= nVertex;
    while (v3 >= nVertex) v3 -= nVertex;
    while (v1 < 0) v1 += nVertex;
    while (v2 < 0) v2 += nVertex;
    while (v3 < 0) v3 += nVertex;

    real tl1 = pTl[n].x;
    real tl2 = pTl[n].y;
    real tl3 = pTl[n].z;

    real s = (real) 0.5*(tl1 + tl2 + tl3);
    real area = sqrt(s*(s - tl1)*(s - tl2)*(s - tl3));

    if (CL == CL_ADVECT) {

      real d1 = pVs[v1];
      real d2 = pVs[v2];
      real d3 = pVs[v3];
      real dG = (d1 + d2 + d3)/three;

      pSource[n] = -dG*area;

      /*
      real f1 = pState[v1];
      real f2 = pState[v2];
      real f3 = pState[v3];

      real dG =
        (real) 0.5*(f1*f1 + f1*f2 + f1*f3 + f2*f2 + f2*f3 + f3*f3)/three;
      pSource[n] = -dG*area;
      */
    }

    if (CL == CL_BURGERS) {
      real d1 = pVs[v1];
      real d2 = pVs[v2];
      real d3 = pVs[v3];
      real dG = (d1 + d2 + d3)/three;

      pSource[n] = -dG*area;

      /*
      real f1 = pState[v1];
      real f2 = pState[v2];
      real f3 = pState[v3];

      //real dG =
      //  (real) 0.5*(f1*f1 + f1*f2 + f1*f3 + f2*f2 + f2*f3 + f3*f3)/three;
      real dG = (f1 + f2 + f3)/three;
      pSource[n] = -dG*area;
      */
    }

  }
}

//######################################################################
/*! \brief Kernel calculating source term contribution to residual.

\param nTriangle Total number of triangles in Mesh
\param problemDef Problem definition
\param nVertex Total number of vertices in Mesh
\param *pTv Pointer to triangle vertices
\param *pTn1 Pointer first triangle edge normal
\param *pTn2 Pointer second triangle edge normal
\param *pTn3 Pointer third triangle edge normal
\param *pTl Pointer to triangle edge lengths
\param *pVp Pointer to external potential at vertices
\param *pState Pointer to state vector
\param *pSource Pointer to source vector (output)
\param *pVs Pointer to vertex source term */
//######################################################################

template<class realNeq, ConservationLaw CL>
__global__ void
devCalcSource(int nTriangle, ProblemDefinition problemDef,
              int nVertex, const int3 *pTv,
              const real2 *pTn1, const real2 *pTn2, const real2 *pTn3,
              const real3 *pTl, const real *pVp,
              const realNeq *pState, realNeq *pSource, const realNeq *pVs)
{
  // n = vertex number
  int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nTriangle) {
    CalcSourceSingle<CL>(n, problemDef, nVertex,
                         pTv, pTn1, pTn2, pTn3,
                         pTl, pVp, pState, pSource, pVs);

    n += blockDim.x*gridDim.x;
  }
}

//#########################################################################
/*! Calculate source contribution to residual. Result will be in \a triangleResidueSource.

\param state State vector to base source term calculation on. */
//#########################################################################

template <class realNeq, ConservationLaw CL>
void Simulation<realNeq, CL>::CalcSource(Array<realNeq> *state)
{
  int nTriangle = mesh->GetNTriangle();
  int nVertex = mesh->GetNVertex();

  ProblemDefinition problemDef = simulationParameter->problemDef;

  const int3 *pTv = mesh->TriangleVerticesData();
  const real *pVp = vertexPotential->GetPointer();
  const realNeq *pState = state->GetPointer();
  realNeq *pVs = vertexSource->GetPointer();
  realNeq *pSource = triangleResidueSource->GetPointer();

  const real2 *pTn1 = mesh->TriangleEdgeNormalsData(0);
  const real2 *pTn2 = mesh->TriangleEdgeNormalsData(1);
  const real2 *pTn3 = mesh->TriangleEdgeNormalsData(2);

  const real3 *pTl = mesh->TriangleEdgeLengthData();
  const real2 *pVc = mesh->VertexCoordinatesData();

  // First calculate sources at vertices
  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devCalcVertexSource<realNeq, CL>,
                                       (size_t) 0, 0);

    devCalcVertexSource<realNeq, CL><<<nBlocks, nThreads>>>
      (nVertex, problemDef, pVc, pVp, pState, pVs);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int i = 0; i < nVertex; i++)
      CalcVertexSourceSingle<CL>(i, problemDef, pVc, pVp, pState, pVs);
  }

  // Average over triangles
  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devCalcSource<realNeq, CL>,
                                       (size_t) 0, 0);

    devCalcSource<realNeq, CL><<<nBlocks, nThreads>>>
      (nTriangle, problemDef, nVertex,
       pTv, pTn1, pTn2, pTn3, pTl, pVp, pState, pSource, pVs);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int i = 0; i < nTriangle; i++)
      CalcSourceSingle<CL>(i, problemDef, nVertex,
                           pTv, pTn1, pTn2, pTn3, pTl,
                           pVp, pState, pSource, pVs);
  }
}

//##############################################################################
// Instantiate
//##############################################################################

template void Simulation<real, CL_ADVECT>::CalcSource(Array<real> *state);
template void Simulation<real, CL_BURGERS>::CalcSource(Array<real> *state);
template void Simulation<real3, CL_CART_ISO>::CalcSource(Array<real3> *state);
template void Simulation<real3, CL_CYL_ISO>::CalcSource(Array<real3> *state);
template void Simulation<real4, CL_CART_EULER>::CalcSource(Array<real4> *state);

}  // namespace astrix
