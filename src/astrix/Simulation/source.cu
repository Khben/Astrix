#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file source.cu
\brief File containing function to calculate source term contribution to residual.

\section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/

#include <iostream>

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "../Mesh/mesh.h"
#include "./simulation.h"
#include "../Common/cudaLow.h"
#include "./Param/simulationparameter.h"

namespace astrix {

//######################################################################
/*! \brief Calculating source term contribution to residual for single triangle.

\param n Triangle to consider
\param problemDef Problem definition
\param nVertex Total number of vertices in Mesh
\param *pTv Pointer to triangle vertices
\param *pTn1 Pointer first triangle edge normal
\param *pTn2 Pointer second triangle edge normal
\param *pTn3 Pointer third triangle edge normal
\param *pTl Pointer to triangle edge lengths
\param *pVp Pointer to external potential at vertices
\param *pState Pointer to state vector
\param *pSource Pointer to source vector (output)  */
//######################################################################

__host__ __device__
void CalcSourceSingle(int n, ProblemDefinition problemDef,
                      int nVertex, const int3 *pTv,
                      const real2 *pTn1, const real2 *pTn2, const real2 *pTn3,
                      const real3 *pTl, const real *pVp,
                      const real4 *pState, real4 *pSource)
{
  pSource[n].x = 0.0;
  pSource[n].y = 0.0;
  pSource[n].z = 0.0;
  pSource[n].w = 0.0;

  if (problemDef == PROBLEM_SOURCE) {
    real three = (real) 3.0;
    real half = (real) 0.5;

    // Vertices belonging to triangle
    int v1 = pTv[n].x;
    int v2 = pTv[n].y;
    int v3 = pTv[n].z;
    while (v1 >= nVertex) v1 -= nVertex;
    while (v2 >= nVertex) v2 -= nVertex;
    while (v3 >= nVertex) v3 -= nVertex;
    while (v1 < 0) v1 += nVertex;
    while (v2 < 0) v2 += nVertex;
    while (v3 < 0) v3 += nVertex;

    real tl1 = pTl[n].x;
    real tl2 = pTl[n].y;
    real tl3 = pTl[n].z;

    real d1 = pState[v1].x;
    real d2 = pState[v2].x;
    real d3 = pState[v3].x;
    real dG = (d1 + d2 + d3)/three;

    /*
    real m1 = pState[v1].y;
    real m2 = pState[v2].y;
    real m3 = pState[v3].y;
    real mG = (m1 + m2 + m3)/three;

    real n1 = pState[v1].z;
    real n2 = pState[v2].z;
    real n3 = pState[v3].z;
    real nG = (n1 + n2 + n3)/three;
    */

    real dpotdx = half*
      (pVp[v1]*pTn1[n].x*tl1 +
       pVp[v2]*pTn2[n].x*tl2 +
       pVp[v3]*pTn3[n].x*tl3);
    real dpotdy = half*
      (pVp[v1]*pTn1[n].y*tl1 +
       pVp[v2]*pTn2[n].y*tl2 +
       pVp[v3]*pTn3[n].y*tl3);

    pSource[n].x = 0.0;
    pSource[n].y = dG*dpotdx;
    pSource[n].z = dG*dpotdy;
    pSource[n].w = 0.0;//mG*dpotdx + nG*dpotdy;
  }
}

__host__ __device__
void CalcSourceSingle(int n, ProblemDefinition problemDef,
                      int nVertex, const int3 *pTv,
                      const real2 *pTn1, const real2 *pTn2, const real2 *pTn3,
                      const real3 *pTl, const real *pVp,
                      const real3 *pState, real3 *pSource)
{
  pSource[n].x = 0.0;
  pSource[n].y = 0.0;
  pSource[n].z = 0.0;

  if (problemDef == PROBLEM_SOURCE) {
    real three = (real) 3.0;
    real half = (real) 0.5;

    // Vertices belonging to triangle
    int v1 = pTv[n].x;
    int v2 = pTv[n].y;
    int v3 = pTv[n].z;
    while (v1 >= nVertex) v1 -= nVertex;
    while (v2 >= nVertex) v2 -= nVertex;
    while (v3 >= nVertex) v3 -= nVertex;
    while (v1 < 0) v1 += nVertex;
    while (v2 < 0) v2 += nVertex;
    while (v3 < 0) v3 += nVertex;

    real tl1 = pTl[n].x;
    real tl2 = pTl[n].y;
    real tl3 = pTl[n].z;

    real d1 = pState[v1].x;
    real d2 = pState[v2].x;
    real d3 = pState[v3].x;
    real dG = (d1 + d2 + d3)/three;

    real dpotdx = half*
      (pVp[v1]*pTn1[n].x*tl1 +
       pVp[v2]*pTn2[n].x*tl2 +
       pVp[v3]*pTn3[n].x*tl3);
    real dpotdy = half*
      (pVp[v1]*pTn1[n].y*tl1 +
       pVp[v2]*pTn2[n].y*tl2 +
       pVp[v3]*pTn3[n].y*tl3);

    pSource[n].x = 0.0;
    pSource[n].y = dG*dpotdx;
    pSource[n].z = dG*dpotdy;
  }
}

__host__ __device__
void CalcSourceSingle(int n, ProblemDefinition problemDef,
                      int nVertex, const int3 *pTv,
                      const real2 *pTn1, const real2 *pTn2, const real2 *pTn3,
                      const real3 *pTl, const real *pVp,
                      const real *pState, real *pSource)
{
  pSource[n] = 0.0;

  if (problemDef == PROBLEM_SOURCE) {
    real three = (real) 3.0;

    // Vertices belonging to triangle
    int v1 = pTv[n].x;
    int v2 = pTv[n].y;
    int v3 = pTv[n].z;
    while (v1 >= nVertex) v1 -= nVertex;
    while (v2 >= nVertex) v2 -= nVertex;
    while (v3 >= nVertex) v3 -= nVertex;
    while (v1 < 0) v1 += nVertex;
    while (v2 < 0) v2 += nVertex;
    while (v3 < 0) v3 += nVertex;

    real tl1 = pTl[n].x;
    real tl2 = pTl[n].y;
    real tl3 = pTl[n].z;

    real d1 = pState[v1];
    real d2 = pState[v2];
    real d3 = pState[v3];
    real dG = (d1 + d2 + d3)/three;

    real s = (real) 0.5*(tl1 + tl2 + tl3);
    real area = sqrt(s*(s - tl1)*(s - tl2)*(s - tl3));

    pSource[n] = dG*area;
  }
}

//######################################################################
/*! \brief Kernel calculating source term contribution to residual.

\param nTriangle Total number of triangles in Mesh
\param problemDef Problem definition
\param nVertex Total number of vertices in Mesh
\param *pTv Pointer to triangle vertices
\param *pTn1 Pointer first triangle edge normal
\param *pTn2 Pointer second triangle edge normal
\param *pTn3 Pointer third triangle edge normal
\param *pTl Pointer to triangle edge lengths
\param *pVp Pointer to external potential at vertices
\param *pState Pointer to state vector
\param *pSource Pointer to source vector (output)  */
//######################################################################

template<class realNeq, ConservationLaw CL>
__global__ void
devCalcSource(int nTriangle, ProblemDefinition problemDef,
              int nVertex, const int3 *pTv,
              const real2 *pTn1, const real2 *pTn2, const real2 *pTn3,
              const real3 *pTl, const real *pVp,
              const realNeq *pState, realNeq *pSource)
{
  // n = vertex number
  int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nTriangle) {
    CalcSourceSingle(n, problemDef, nVertex,
                     pTv, pTn1, pTn2, pTn3,
                     pTl, pVp, pState, pSource);

    n += blockDim.x*gridDim.x;
  }
}

//#########################################################################
/*! Calculate source contribution to residual. Result will be in \a triangleResidueSource.

\param state State vector to base source term calculation on. */
//#########################################################################

template <class realNeq, ConservationLaw CL>
void Simulation<realNeq, CL>::CalcSource(Array<realNeq> *state)
{
  int nTriangle = mesh->GetNTriangle();
  int nVertex = mesh->GetNVertex();

  ProblemDefinition problemDef = simulationParameter->problemDef;

  const int3 *pTv = mesh->TriangleVerticesData();
  const real *pVp = vertexPotential->GetPointer();
  const realNeq *pState = state->GetPointer();
  realNeq *pSource = triangleResidueSource->GetPointer();

  const real2 *pTn1 = mesh->TriangleEdgeNormalsData(0);
  const real2 *pTn2 = mesh->TriangleEdgeNormalsData(1);
  const real2 *pTn3 = mesh->TriangleEdgeNormalsData(2);

  const real3 *pTl = mesh->TriangleEdgeLengthData();

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devCalcSource<realNeq, CL>,
                                       (size_t) 0, 0);

    devCalcSource<realNeq, CL><<<nBlocks, nThreads>>>
      (nTriangle, problemDef, nVertex,
       pTv, pTn1, pTn2, pTn3, pTl, pVp, pState, pSource);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int i = 0; i < nTriangle; i++)
      CalcSourceSingle(i, problemDef, nVertex,
                       pTv, pTn1, pTn2, pTn3, pTl,
                       pVp, pState, pSource);
  }
}

//##############################################################################
// Instantiate
//##############################################################################

template void Simulation<real, CL_ADVECT>::CalcSource(Array<real> *state);
template void Simulation<real, CL_BURGERS>::CalcSource(Array<real> *state);
template void Simulation<real3, CL_CART_ISO>::CalcSource(Array<real3> *state);
template void Simulation<real4, CL_CART_EULER>::CalcSource(Array<real4> *state);

}  // namespace astrix
