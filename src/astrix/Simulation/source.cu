#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file source.cu
\brief File containing function to calculate source term contribution to residual.

\section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/

#include <iostream>

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "../Mesh/mesh.h"
#include "./simulation.h"
#include "../Common/cudaLow.h"
#include "./Param/simulationparameter.h"

namespace astrix {

//######################################################################
//######################################################################

__host__ __device__
void CalcSourceSingle(int n, ProblemDefinition problemDef, int nVertex,
                      const int3 *pTv, const real2 *pVc,
                      const real2 *pTn1, const real2 *pTn2, const real2 *pTn3,
                      const real3 *pTl, const real *pVp,
                      const real4 *pState, real4 *pSource)
{
  pSource[n].x = 0.0;
  pSource[n].y = 0.0;
  pSource[n].z = 0.0;
  pSource[n].w = 0.0;

  if (problemDef == PROBLEM_SOURCE) {
    real three = (real) 3.0;

    // Vertices belonging to triangle
    int v1 = pTv[n].x;
    int v2 = pTv[n].y;
    int v3 = pTv[n].z;
    while (v1 >= nVertex) v1 -= nVertex;
    while (v2 >= nVertex) v2 -= nVertex;
    while (v3 >= nVertex) v3 -= nVertex;
    while (v1 < 0) v1 += nVertex;
    while (v2 < 0) v2 += nVertex;
    while (v3 < 0) v3 += nVertex;

    real tl1 = pTl[n].x;
    real tl2 = pTl[n].y;
    real tl3 = pTl[n].z;

    real d1 = pState[v1].x;
    real d2 = pState[v2].x;
    real d3 = pState[v3].x;
    real dG = (d1 + d2 + d3)/three;

    real m1 = pState[v1].z;
    real m2 = pState[v2].z;
    real m3 = pState[v3].z;
    real mG = (m1 + m2 + m3)/three;

    real s = (real) 0.5*(tl1 + tl2 + tl3);
    real area = sqrt(s*(s - tl1)*(s - tl2)*(s - tl3));

    pSource[n].x = 0.0;
    pSource[n].y = 0.0;
    pSource[n].z = dG*0.1*area;
    pSource[n].w = mG*0.1*area;
  }
}

__host__ __device__
void CalcSourceSingle(int n, ProblemDefinition problemDef, int nVertex,
                      const int3 *pTv, const real2 *pVc,
                      const real2 *pTn1, const real2 *pTn2, const real2 *pTn3,
                      const real3 *pTl, const real *pVp,
                      const real3 *pState, real3 *pSource)
{
  pSource[n].x = 0.0;
  pSource[n].y = 0.0;
  pSource[n].z = 0.0;

  if (problemDef == PROBLEM_SOURCE) {
    real three = (real) 3.0;

    // Vertices belonging to triangle
    int v1 = pTv[n].x;
    int v2 = pTv[n].y;
    int v3 = pTv[n].z;
    while (v1 >= nVertex) v1 -= nVertex;
    while (v2 >= nVertex) v2 -= nVertex;
    while (v3 >= nVertex) v3 -= nVertex;
    while (v1 < 0) v1 += nVertex;
    while (v2 < 0) v2 += nVertex;
    while (v3 < 0) v3 += nVertex;

    real tl1 = pTl[n].x;
    real tl2 = pTl[n].y;
    real tl3 = pTl[n].z;

    real d1 = pState[v1].x;
    real d2 = pState[v2].x;
    real d3 = pState[v3].x;
    real dG = (d1 + d2 + d3)/three;

    real s = (real) 0.5*(tl1 + tl2 + tl3);
    real area = sqrt(s*(s - tl1)*(s - tl2)*(s - tl3));

    pSource[n].x = 0.0;
    pSource[n].y = 0.0;
    pSource[n].z = dG*0.1*area;
  }
}

__host__ __device__
void CalcSourceSingle(int n, ProblemDefinition problemDef, int nVertex,
                      const int3 *pTv, const real2 *pVc,
                      const real2 *pTn1, const real2 *pTn2, const real2 *pTn3,
                      const real3 *pTl, const real *pVp,
                      const real *pState, real *pSource)
{
  pSource[n] = 0.0;

  if (problemDef == PROBLEM_SOURCE) {
    real three = (real) 3.0;

    // Vertices belonging to triangle
    int v1 = pTv[n].x;
    int v2 = pTv[n].y;
    int v3 = pTv[n].z;
    while (v1 >= nVertex) v1 -= nVertex;
    while (v2 >= nVertex) v2 -= nVertex;
    while (v3 >= nVertex) v3 -= nVertex;
    while (v1 < 0) v1 += nVertex;
    while (v2 < 0) v2 += nVertex;
    while (v3 < 0) v3 += nVertex;

    real tl1 = pTl[n].x;
    real tl2 = pTl[n].y;
    real tl3 = pTl[n].z;

    real d1 = pState[v1];
    real d2 = pState[v2];
    real d3 = pState[v3];
    real dG = (d1 + d2 + d3)/three;

    real s = (real) 0.5*(tl1 + tl2 + tl3);
    real area = sqrt(s*(s - tl1)*(s - tl2)*(s - tl3));

    pSource[n] = dG*area;

    /*
    real two = (real) 2.0;
    real three = (real) 3.0;

    // Vertices belonging to triangle
    int v1 = pTv[n].x;
    int v2 = pTv[n].y;
    int v3 = pTv[n].z;
    while (v1 >= nVertex) v1 -= nVertex;
    while (v2 >= nVertex) v2 -= nVertex;
    while (v3 >= nVertex) v3 -= nVertex;
    while (v1 < 0) v1 += nVertex;
    while (v2 < 0) v2 += nVertex;
    while (v3 < 0) v3 += nVertex;

    real tl1 = pTl[n].x;
    real tl2 = pTl[n].y;
    real tl3 = pTl[n].z;

    real x1 = pVc[v1].x;
    real x2 = pVc[v2].x;
    real x3 = pVc[v3].x;
    real xG = (x1 + x2 + x3)/three;

    real q1 = pState[v1];
    real q2 = pState[v2];
    real q3 = pState[v3];
    real qG = (q1 + q2 + q3)/three;

    real s = (real) 0.5*(tl1 + tl2 + tl3);
    real area = sqrt(s*(s - tl1)*(s - tl2)*(s - tl3));

    //real s1 = -two*x1*q1;
    //real s2 = -two*x2*q2;
    //real s3 = -two*x3*q3;

    //pSource[n] = -area*10.0*(s1 + s2 + s3)/three;
    pSource[n] = -area*10.0*(-two*xG*qG);
    //pSource[n] = -area*qG;
    */
  }
}

//######################################################################
//######################################################################

__global__ void
devCalcSource(int nTriangle, ProblemDefinition problemDef, int nVertex,
              const int3 *pTv, const real2 *pVc,
              const real2 *pTn1, const real2 *pTn2, const real2 *pTn3,
              const real3 *pTl, const real *pVp,
              const realNeq *pState, realNeq *pSource)
{
  // n = vertex number
  int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nTriangle) {
    CalcSourceSingle(n, problemDef, nVertex,
                     pTv, pVc, pTn1, pTn2, pTn3,
                     pTl, pVp, pState, pSource);

    n += blockDim.x*gridDim.x;
  }
}

//#########################################################################
//#########################################################################

void Simulation::CalcSource(Array<realNeq> *state)
{
  int nTriangle = mesh->GetNTriangle();
  int nVertex = mesh->GetNVertex();

  ProblemDefinition problemDef = simulationParameter->problemDef;

  const int3 *pTv = mesh->TriangleVerticesData();
  const real *pVp = vertexPotential->GetPointer();
  const real2 *pVc = mesh->VertexCoordinatesData();
  const realNeq *pState = state->GetPointer();
  realNeq *pSource = triangleResidueSource->GetPointer();

  const real2 *pTn1 = mesh->TriangleEdgeNormalsData(0);
  const real2 *pTn2 = mesh->TriangleEdgeNormalsData(1);
  const real2 *pTn3 = mesh->TriangleEdgeNormalsData(2);

  const real3 *pTl = mesh->TriangleEdgeLengthData();

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devCalcSource,
                                       (size_t) 0, 0);

    devCalcSource<<<nBlocks, nThreads>>>
      (nTriangle, problemDef, nVertex,
       pTv, pVc, pTn1, pTn2, pTn3, pTl, pVp, pState, pSource);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int i = 0; i < nTriangle; i++)
      CalcSourceSingle(i, problemDef, nVertex,
                       pTv, pVc, pTn1, pTn2, pTn3, pTl,
                       pVp, pState, pSource);
  }
}

}  // namespace astrix
