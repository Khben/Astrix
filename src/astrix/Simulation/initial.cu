#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file initial.cu
\brief Functions to set initial conditions

*/ /* \section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/
#include <iostream>

#include <gsl/gsl_sf_bessel.h>
#include <boost/math/special_functions/bessel.hpp>

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "../Mesh/mesh.h"
#include "./simulation.h"
#include "../Common/cudaLow.h"
#include "../Common/inlineMath.h"
#include "./Param/simulationparameter.h"

namespace astrix {

//! Bump helper function
__host__ __device__
real funcFbump(real t)
{
  if (t <= 0.0f) return (real) 0.0;
  return exp(-(real) 1.0/(t + 1.0e-10));
}

//! Bump main function
__host__ __device__
real funcBump(real t)
{
  return funcFbump(t)/(funcFbump(t) + funcFbump((real) 1.0 - t));
}

//##############################################################################
/*! \brief Set initial conditions at vertex \a n

\param n Vertex to consider
\param *pVc Pointer to coordinates of vertices
\param problemDef Problem definition
\param *pVpot Pointer to gravitational potential at vertices
\param *state Pointer to state vector (output)
\param G Ratio of specific heats
\param time Simulation time
\param Px Length of x domain
\param Py Length of y domain
\param denspow Density power law index (cylindrical isothermal only)
\param cs0 Soundspeed at x = 0 (cylindrical isothermal only)
\param cspow Soundspeed power law index (cylindrical isothermal only)*/
//##############################################################################

template <ConservationLaw CL>
__host__ __device__
void SetInitialSingle(int n, const real2 *pVc, ProblemDefinition problemDef,
                      real *pVpot, real4 *state, real G, real time,
                      real Px, real Py, real denspow, real cs0, real cspow)
{
  // const real onethird = (real) (1.0/3.0);
  const real zero = (real) 0.0;
  const real half = (real) 0.5;
  const real one = (real) 1.0;
  const real two = (real) 2.0;
  const real five = (real) 5.0;

  real vertX = pVc[n].x;
  real vertY = pVc[n].y;

  real dens = zero;
  real momx = zero;
  real momy = zero;
  real ener = zero;

  if (problemDef == PROBLEM_GAUSS) {
    real x = vertX;
    //real y = vertY;

    real X_VELOCITY = 2.0, pres = 100.0;
    real CENTRE = 0.2;
    real S,W,RHO_0 = 10.0,RHO_PULSE = 50.0;

    S = abs(CENTRE - x);
    W = 0.1;

    dens = RHO_PULSE*exp(-S*S/(W*W)) + RHO_0*(1.0-exp(-S*S/(W*W)));
    momx = dens*X_VELOCITY;
    momy = dens*0.0000000001;

    ener = half*(Sq(momx) + Sq(momy))/dens + pres/(G - one);
  }

  if (problemDef == PROBLEM_LINEAR) {
    real amp = (real) 1.0e-4;
    real k = two*M_PI;
    real c0 = one;
    real p0 = c0*c0/G;

    dens = one;
    momx = zero;
    momy = (real) 1.0e-10;

    dens += amp*cos(k*vertX);
    momx += amp*cos(k*vertX);
    p0   += amp*cos(k*vertX);

    ener = half*(Sq(momx) + Sq(momy))/dens + p0/(G - one);
  }

  if (problemDef == PROBLEM_VORTEX) {
    real x = vertX;
    real y = vertY;
    real vx = 1.0;
    real vy = 0.0;

    // Temperature
    real temp = 1.0;

    //for (int i = -1; i < 2; i++) {
    //real xc = five + time + i*Px;
      real xc = five + vx*time;
      real yc = five;

      real beta = five;

      real r = sqrt(Sq(x - xc)+Sq(y - yc)) + (real) 1.0e-10;

      temp -= (G - one)*Sq(beta)*exp(one - r*r)/(8.0f*G*M_PI*M_PI);

      vx -= half*(y - yc)*beta*exp(half - half*r*r)/M_PI;
      vy += half*(x - xc)*beta*exp(half - half*r*r)/M_PI;
      //}

    dens = std::pow(temp, (real)(one/(G - one)));
    real pres = dens*temp;

    momx = (real)1.0e-10 + dens*vx;
    momy = (real)2.0e-10 + dens*vy;
    ener = half*(Sq(momx) + Sq(momy))/dens + pres/(G - one);
  }

  if (problemDef == PROBLEM_CYL) {
    dens = 1.0;
    momx = 2.0;
    momy = 1.0e-10;
    ener = half*(Sq(momx) + Sq(momy))/dens + 2.5f/(G - one);
  }

  if (problemDef == PROBLEM_KH) {
    real a = 4.0f;
    real smoothHat =
      funcBump(a*(vertY - 0.25f) + 0.5f)*funcBump(-a*(vertY - 0.75f) + 0.5f);

    dens = 1.0f + smoothHat;
    momx = dens*(-0.5f + smoothHat);
    momy = 0.0f;
    ener = 0.5f*(Sq(momx)+Sq(momy))/dens + 2.5f/(G - 1.0f);
  }

  if (problemDef == PROBLEM_SOD) {
    /*
    real v = 0.5;
    dens = 1.0;
    if (vertX > 0.25 + v*time)
      dens = 0.5;
    momx = dens*v;
    momy = 0.0;
    ener = 0.5f*(Sq(momx)+Sq(momy))/dens + 1.0f/(G - 1.0f);
    */

    /*
    real v = sqrt(G/(3.0 - G));
    if (vertX > 0.0) v = -v;

    dens = 1.0;
    momx = v;
    momy = 0.0;
    ener = 0.5f*(Sq(momx)+Sq(momy))/dens + 1.0f/(G - 1.0f);

    if (abs(vertX) < abs(v)*time) {
      dens = 2.0;
      momx = 0.0;
      momy = 0.0;
      ener = (1.0f + dens*v*v)/(G - 1.0f);
    }
    */


    // Left state
    real dL = 1.0;
    real pL = 1.0;
    real cL = sqrt(G*pL/dL);

    // Right state
    real dR = 0.125;
    real pR = 0.1;
    real cR = sqrt(G*pR/dR);

    real G1 = G - 1.0;
    real m = G1/(G + 1.0);

    real pmid = 0.30313017805;

    real vpost = 2.0*sqrt(G)*(1.0 - std::pow(pmid, 0.5*G1/G))/G1;

    real dmidR = dR*(pmid + m*pR)/(pR + m*pmid);
    real dmidL = dL*std::pow(pmid/pL, 1.0/G);

    real umid = (pmid - pR)/sqrt(0.5*dR*((G + 1.0)*pmid + G1*pR));
    real vshock = dmidR*umid/(dmidR - dR);

    real x0 = 0.5;
    real x1 = x0 - cL*time;
    real x2 = x0 - cL*time*(std::pow(dmidL/dL, 0.5*G1) + m - 1.0)/m;
    real x3 = x0 + vpost*time;
    real x4 = x0 + vshock*time;

    dens = dL;
    momx = 1.0e-10f;
    momy = 1.0e-10f;
    real pres = pL;

    if (vertX > x1) {
      real c = m*(x0 - vertX)/time + (1.0 - m)*cL;
      dens = dL*std::pow(c/cL, 2.0/G1);
      momx = dens*((vertX - x1)*umid/(x2 - x1));
      momy = 1.0e-10f;
      pres = pL*std::pow(dens/dL, G);
    }
    if (vertX > x2) {
      dens = dmidL;
      momx = dens*umid;
      momy = 1.0e-10f;
      pres = pmid;
    }
    if (vertX > x3) {
      dens = dmidR;
      momx = dens*umid;
      momy = 1.0e-10f;
      pres = pmid;
    }
    if (vertX > x4) {
      dens = dR;
      momx = 1.0e-10f;
      momy = 1.0e-10f;
      pres = pR;
    }

    ener = 0.5f*(Sq(momx)+Sq(momy))/dens + pres/G1;


  }

  if (problemDef == PROBLEM_NOH) {
    real x = vertX;
    real y = vertY;
    real r = sqrt(x*x + y*y);

    real pres = 1.0e-6;

    if (r < time/3.0) {
      dens = 16.0;
      momx = zero;
      momy = zero;
      pres = 16.0/3.0;
    } else {
      dens = one + time/(r + 1.0e-10);
      momx = dens*(-x/(r + 1.0e-10) + 1.0e-10);
      momy = dens*(-y/(r + 1.0e-10) - 2.0e-10);
    }

    ener = half*(Sq(momx) + Sq(momy))/dens + pres/(G - one);
  }

  if (problemDef == PROBLEM_BLAST) {
    // Interacting blast waves
    real p = 0.01f;
    if (vertX < 0.1f) p = 1000.0f;
    if (vertX > 0.9f) p = 100.0f;

    dens = 1.0f;
    momx = 1.0e-10f;
    momy = 1.0e-10f;
    ener = 0.5f*(Sq(momx)+Sq(momy))/dens + p/(G - 1.0f);
  }

  if (problemDef == PROBLEM_RIEMANN) {
    real f = 1.0f - (vertX > 0.8f);
    real g = 1.0f - (vertY > 0.8f);

    // CASE 3
    real densRB = 1.5f;
    real momxRB = 1.0e-10f;
    real momyRB = 1.0e-10f;
    real enerRB = 0.5f*(Sq(momxRB) + Sq(momyRB))/densRB + 1.5f/(G - 1.0f);

    real densLB = 0.5322581f;
    real momxLB = 1.2060454f*densLB;
    real momyLB = 0.0f;
    real enerLB = 0.5f*(Sq(momxLB) + Sq(momyLB))/densLB + 0.3f/(G - 1.0f);

    real densRO = 0.5322581f;
    real momxRO = 0.0f;
    real momyRO = 1.2060454f*densRO;
    real enerRO = 0.5f*(Sq(momxRO) + Sq(momyRO))/densRO + 0.3f/(G - 1.0f);

    real densLO = 0.1379928f;
    real momxLO = 1.2060454f*densLO;
    real momyLO = 1.2060454f*densLO;
    real enerLO = 0.5f*(Sq(momxLO) + Sq(momyLO))/densLO + 0.0290323f/(G - 1.0f);

    /*
    // CASE 6
    real densRB = 1.0f;
    real momxRB = 0.75f*densRB;
    real momyRB = -0.5f*densRB;
    real enerRB = 0.5f*(Sq(momxRB)+Sq(momyRB))/densRB + 1.0f/(G-1.0f);

    real densLB = 2.0f;
    real momxLB = 0.75f*densLB;
    real momyLB = 0.5f*densLB;
    real enerLB = 0.5f*(Sq(momxLB)+Sq(momyLB))/densLB + 1.0f/(G-1.0f);

    real densRO = 3.0f;
    real momxRO = -0.75f*densRO;
    real momyRO = -0.5f*densRO;
    real enerRO = 0.5f*(Sq(momxRO)+Sq(momyRO))/densRO + 1.0f/(G-1.0f);

    real densLO = 1.0f;
    real momxLO = -0.75f*densLO;
    real momyLO = 0.5f*densLO;
    real enerLO = 0.5f*(Sq(momxLO)+Sq(momyLO))/densLO + 1.0f/(G-1.0f);
    */
    /*
    // CASE 15
    real densRB = 1.0f;
    real momxRB = 0.1f*densRB;
    real momyRB = -0.3f*densRB;
    real enerRB = 0.5f*(Sq(momxRB)+Sq(momyRB))/densRB + 1.0f/(G-1.0f);

    real densLB = 0.5197f;
    real momxLB = -0.6259f*densLB;
    real momyLB = -0.3f*densLB;
    real enerLB = 0.5f*(Sq(momxLB)+Sq(momyLB))/densLB + 0.4f/(G-1.0f);

    real densRO = 0.5313f;
    real momxRO = 0.1f*densRO;
    real momyRO = 0.4276f*densRO;
    real enerRO = 0.5f*(Sq(momxRO)+Sq(momyRO))/densRO + 0.4f/(G-1.0f);

    real densLO = 0.8f;
    real momxLO = 0.1f*densLO;
    real momyLO = -0.3f*densLO;
    real enerLO = 0.5f*(Sq(momxLO)+Sq(momyLO))/densLO + 0.4f/(G-1.0f);
    */
    /*
    // Case 17
    real densRB = 1.0f;
    real momxRB = 1.0e-10f*densRB;
    real momyRB = -0.4f*densRB;
    real enerRB = 0.5f*(Sq(momxRB)+Sq(momyRB))/densRB + 1.0f/(G-1.0f);

    real densLB = 2.0f;
    real momxLB = 1.0e-10f*densLB;
    real momyLB = -0.3f*densLB;
    real enerLB = 0.5f*(Sq(momxLB)+Sq(momyLB))/densLB + 1.0f/(G-1.0f);

    real densRO = 0.5197f;
    real momxRO = 1.0e-10f*densRO;
    real momyRO = -1.1259f*densRO;
    real enerRO = 0.5f*(Sq(momxRO)+Sq(momyRO))/densRO + 0.4f/(G-1.0f);

    real densLO = 1.0625f;
    real momxLO = 1.0e-10f*densLO;
    real momyLO = 0.2145f*densLO;
    real enerLO = 0.5f*(Sq(momxLO)+Sq(momyLO))/densLO + 0.4f/(G-1.0f);
    */

    dens =
      densRB*(1.0f - f)*(1.0f - g) + densLB*f*(1.0f - g) +
      densRO*(1.0f - f)*g + densLO*f*g;
    momx =
      momxRB*(1.0f - f)*(1.0f - g) + momxLB*f*(1.0f - g) +
      momxRO*(1.0f - f)*g + momxLO*f*g;
    momy =
      momyRB*(1.0f - f)*(1.0f - g) + momyLB*f*(1.0f - g) +
      momyRO*(1.0f - f)*g + momyLO*f*g;
    ener =
      enerRB*(1.0f - f)*(1.0f - g) + enerLB*f*(1.0f - g) +
      enerRO*(1.0f - f)*g + enerLO*f*g;
  }

  if (problemDef == PROBLEM_SOURCE) {
    real rhoBot = 1.0;
    real rhoTop = 2.0;
    real Ly = 1.0;
    dens = rhoBot + 0.5*(rhoTop - rhoBot)*(vertY + Ly)/Ly;
    momx = 1.0e-10;
    momy = 0.0;
    real pres = 1.0 - 0.1*(0.5*(rhoTop + rhoBot)*vertY +
                           0.25*vertY*vertY*(rhoTop - rhoBot)/Ly);
    ener = half*(Sq(momx) + Sq(momy))/dens + dens*pVpot[n] + pres/(G - one);

    /*
    real x = vertX;
    real y = vertY;

    real xc = five;
    real yc = five;

    real beta = five;

    real r = sqrt(Sq(x - xc)+Sq(y - yc)) + (real) 1.0e-10;

    real vx = -half*(y - yc)*beta*exp(half - half*r*r)/M_PI;
    real vy = half*(x - xc)*beta*exp(half - half*r*r)/M_PI;

    dens = 1.0;
    momx = (real)1.0e-10 + dens*vx;
    momy = (real)2.0e-10 + dens*vy;
    real pres = 1.0;
    ener = half*(Sq(momx) + Sq(momy))/dens + dens*pVpot[n] + pres/(G - one);
    */

    /*
    real x = vertX;
    real y = vertY;

    real xc = five;
    real yc = five;

    real beta = five;

    real r = sqrt(Sq(x - xc)+Sq(y - yc)) + (real) 1.0e-10;

    real pot = y;//-0.125*beta*beta*exp(1.0 - r*r)/(M_PI*M_PI);

    dens = std::pow(1.0 - (G - 1.0)*pot/G, 1.0/(G - 1.0));
    momx = 1.0e-10;
    momy = 0.0;
    real pres = std::pow(dens, G);
    ener = half*(Sq(momx) + Sq(momy))/dens + dens*pVpot[n] + pres/(G - one);
    */
  }

  state[n].x = dens;
  state[n].y = momx;
  state[n].z = momy;
  state[n].w = ener;
}

//! Version for three equations
template <ConservationLaw CL>
__host__ __device__
void SetInitialSingle(int n, const real2 *pVc, ProblemDefinition problemDef,
                      real *pVpot, real3 *state, real G, real time,
                      real Px, real Py, real denspow, real cs0, real cspow)
{
  real zero = (real) 0.0;
  real one = (real) 1.0;
  real five = (real) 5.0;
  real half = (real) 0.5;

  real vertX = pVc[n].x;
  real vertY = pVc[n].y;

  real dens = (real) 1.0;
  real momx = (real) 0.0;
  real momy = (real) 0.0;

  if (problemDef == PROBLEM_VORTEX) {
    real x = vertX;
    real y = vertY;
    real vx = one;
    real vy = zero;

    real xc = five + vx*time;
    real yc = five;

    real beta = five;

    real r = sqrt(Sq(x - xc)+Sq(y - yc)) + (real) 1.0e-10;

    dens = exp(-Sq(beta/M_PI)*exp(one - r*r)/8.0);
    vx -= half*(y - yc)*beta*exp(half - half*r*r)/M_PI;
    vy += half*(x - xc)*beta*exp(half - half*r*r)/M_PI;

    momx = (real)1.0e-10 + dens*vx;
    momy = (real)2.0e-10 + dens*vy;
  }

  if (problemDef == PROBLEM_SOURCE) {
    //dens = exp(-0.1*vertY);

    real x = vertX;
    real y = vertY;

    real xc = five;
    real yc = five;

    real beta = five;

    real r = sqrt(Sq(x - xc)+Sq(y - yc)) + (real) 1.0e-10;

    real vx = -half*(y - yc)*beta*exp(half - half*r*r)/M_PI;
    real vy = half*(x - xc)*beta*exp(half - half*r*r)/M_PI;

    momx = (real)1.0e-10 + dens*vx;
    momy = (real)2.0e-10 + dens*vy;
  }

  if (problemDef == PROBLEM_DISC) {
    real x = vertX;
    real y = vertY;

    // Cylindrical radius
    real r = exp(x);

    dens = pow(r, denspow);
    real vx = 0.0e-10;
    // Angular momentum
    real vy = sqrt(1.0*r +
                   (denspow + 2.0*cspow)*cs0*cs0*pow(r, 4.0 + 2.0*cspow));

    // Linear perturbations
    real A = 1.0e-5;            // Amplitude
    // Surface density \propto r^{-a}
    real a = 2.0 - denspow;

    /*
    // NO GRAVITY
    real k = 4.0*M_PI/Px;
    real w = 0.5*sqrt(4.0*k*k + a*a - 20.0*a + 36.0)*cs0;
    real f = A*sin(k*x);
    real df = A*k*cos(k*x);

    real u1 = f*cos(w*time)/sqrt(dens);
    real d1 = -sqrt(dens)*(df + (1.0 - 0.5*a)*f)*sin(w*time)/w;
    real v1 = -r*r*f*sin(w*time)*2.0*sqrt(2.0-a)*cs0/(w*sqrt(dens));

    */

    /*
    // GRAVITY: needs denspow=0, soundspeed0=1/sqrt(3), soundspeedPower=-1.5
    real b0 = 5.40278878940795; //1.3196250078464;
    real c1 = 2.6674287565997314; //66.561534022766;
    real w = cs0*b0;

    real q = 2.0*b0*r*sqrt(r)/3.0;
    real g = A*sqrt(r)*(gsl_sf_bessel_J0(q) + c1*gsl_sf_bessel_Y0(q));
    real dg = 0.5*g/r - A*b0*r*(gsl_sf_bessel_J1(q) + c1*gsl_sf_bessel_Y1(q));

    real u1 = g*cos(w*time)/sqrt(dens*r);
    real d1 = sqrt(dens/r)*(0.5*(a - 1.0)*g - r*dg)*sin(w*time)/w;
    //real v1 = -0.5*g*sqrt(1.0 - (a + 1.0)*cs0*cs0)*sin(w*time)/(w*sqrt(dens));
    real v1 = 0.0;
    */

    // DISC: needs denspow=0, soundspeed0=0.1, soundspeedPower=-1.5
    real b0 = 20.829436532817862;
    real c1 = 2.831729237113438;
    real w = cs0*b0;

    real nu = sqrt(1.0 + 4.0/(cs0*cs0) - 8.0*a + a*a - 1.0)/3.0;
    real q = 2.0*b0*r*sqrt(r)/3.0;

    real Jplus = boost::math::cyl_bessel_j(nu, q);
    real Jmin  = boost::math::cyl_bessel_j(-nu, q);
    real dJplus = 0.5*(boost::math::cyl_bessel_j(nu - 1.0, q) -
                       boost::math::cyl_bessel_j(nu + 1.0, q));
    real dJmin  = 0.5*(boost::math::cyl_bessel_j(-nu - 1.0, q) -
                       boost::math::cyl_bessel_j(-nu + 1.0, q));

    real g = A*sqrt(r)*(Jmin + c1*Jplus);

    real dg = 0.5*g/r + A*b0*r*(dJmin + c1*dJplus);

    real u1 = g*cos(w*time)/sqrt(dens*r);
    real d1 = sqrt(dens/r)*(0.5*(a - 1.0)*g - r*dg)*sin(w*time)/w;
    real v1 = -0.5*g*sqrt(1.0 - (a + 1.0)*cs0*cs0)*sin(w*time)/(w*sqrt(dens));
    //real v1 = 0.0;

    /*
#ifndef __CUDA_ARCH__
    std::cout << "Period: " << 2.0*M_PI/w << std::endl;
    std::cout << u1 << " " << d1 << " " << v1 << std::endl;
    //std::cout << vy << std::endl;
    //std::cout << r + (denspow + 2.0*cspow)*cs0*cs0*pow(r, 4.0 + 2.0*cspow)
    //          << std::endl;
    int qq; std::cin >> qq;
#endif
    */

    dens += d1;
    vx += u1;
    vy += v1;


    //if (r > 0.5 && r < 2.4)
    //  dens *= (1.0 + 0.1*exp(-Sq(y - 3.14)/0.01));
    momx = dens*vx;
    momy = dens*vy;
  }

  state[n].x = dens;
  state[n].y = momx;
  state[n].z = momy;
}

//! Version for single equation
template <ConservationLaw CL>
__host__ __device__
void SetInitialSingle(int n, const real2 *pVc, ProblemDefinition problemDef,
                      real *pVpot, real *state, real G, real time,
                      real Px, real Py, real denspow, real cs0, real cspow)
{
  real vertX = pVc[n].x;
  real vertY = pVc[n].y;

  real dens = (real) 1.0;

  if (problemDef == PROBLEM_LINEAR) {
    for (int i = -1; i < 2; i++) {
      real x = vertX;
      real xc = time + i*Px;

      if (fabs(x - xc) <= (real) 0.25) dens += Sq(cos(2.0*M_PI*(x - xc)));
    }
  }

  if (problemDef == PROBLEM_VORTEX) {
    real half = (real) 0.5;

    for (int i = -1; i < 2; i++) {
      real x = vertX;
      real y = vertY;

      real xc = half + i*Px + time;
      real yc = half;

      real r = sqrt(Sq(x - xc) + Sq(y - yc));

      if (r <= (real) 0.25) dens += Sq(cos(2.0*M_PI*r));
    }
  }

  if (problemDef == PROBLEM_RIEMANN) {
    real x = vertX;
    real y = vertY;

    dens = (real) 1.0e-10;
    if (x  > (real) -0.6 && x < (real) - 0.1 &&
        y > (real) -0.35 && y < (real) 0.15) dens += 1.0;
  }

  if (problemDef == PROBLEM_SOURCE) {
    if (CL == CL_ADVECT) {
      // Source = u, periodic
      //dens = cos(2.0*M_PI*(vertX - time))*exp(time);

      // Source = u, not periodic
      //dens = exp(0.5*(vertX + time));

      // Source = u^2, not periodic
      dens = -(real) 2.0/(vertX + time + 2.0);
    }
    if (CL == CL_BURGERS) {
      // Source = u
      //dens = vertX/((real) 1.0 + exp(-time));
      //dens = 0.5*exp(2.0*time)*(sqrt(1.0 + 4.0*vertX*exp(-2.0*time)) - 1.0);

      // Source = u^2
      dens = -((real) 1.0 + exp(vertX))/((real) 1.0 + time);
    }
  }

  state[n] = dens;
}

//######################################################################
/*! \brief Kernel setting initial conditions

\param nVertex Total number of vertices in Mesh
\param *pVc Pointer to coordinates of vertices
\param problemDef Problem definition
\param *pVertexPotential Pointer to gravitational potential at vertices
\param *state Pointer to state vector (output)
\param G Ratio of specific heats
\param time Current simulation time
\param Px Length of x domain
\param Py Length of y domain
\param denspow Density power law index (cylindrical isothermal only)
\param cs0 Soundspeed at x = 0 (cylindrical isothermal only)
\param cspow Soundspeed power law index (cylindrical isothermal only)
\param *pVertexFlag Pointer to flags indicating whether vertex is part of boundary
\param boundaryFlag Flag whether to set only boundary vertices*/
//######################################################################

template<class realNeq, ConservationLaw CL>
__global__ void
devSetInitial(int nVertex, const real2 *pVc, ProblemDefinition problemDef,
              real *pVertexPotential, realNeq *state,
              real G, real time, real Px, real Py,
              real denspow, real cs0, real cspow,
              const int *pVertexFlag, int boundaryFlag)
{
  // n = vertex number
  int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nVertex) {
    // Do only specific vertices
    int do_flag = 1;
    if (boundaryFlag != 0)
      if (pVertexFlag[n] == 0) do_flag = 0;

    if (do_flag)
      SetInitialSingle<CL>(n, pVc, problemDef, pVertexPotential, state,
                           G, time, Px, Py, denspow, cs0, cspow);

    n += blockDim.x*gridDim.x;
  }
}

//######################################################################
/*! Set initial conditions for all vertices based on problemSpecification

\param time Current simulation time, to get exact solution at this time (if available)
\param boundaryFlag Flag whether to set only boundary vertices*/
//######################################################################

template <class realNeq, ConservationLaw CL>
void Simulation<realNeq, CL>::SetInitial(real time,
                                         int boundaryFlag)
{
  int nVertex = mesh->GetNVertex();

  realNeq *state = vertexState->GetPointer();
  real *pVertexPotential = vertexPotential->GetPointer();
  const real2 *pVc = mesh->VertexCoordinatesData();

  real Px = mesh->GetPx();
  real Py = mesh->GetPy();
  real G = simulationParameter->specificHeatRatio;
  ProblemDefinition p = simulationParameter->problemDef;

  const real denspow = simulationParameter->densityPower;
  const real cs0 = simulationParameter->soundspeed0;
  const real cspow = simulationParameter->soundspeedPower;

  const int *pVertexFlag = mesh->VertexBoundaryFlagData();

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devSetInitial<realNeq, CL>,
                                       (size_t) 0, 0);

    devSetInitial<realNeq, CL><<<nBlocks, nThreads>>>
      (nVertex, pVc, p, pVertexPotential, state, G, time, Px, Py,
       denspow, cs0, cspow,
       pVertexFlag, boundaryFlag);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int n = 0; n < nVertex; n++) {
      // Do only specific vertices
      int do_flag = 1;
      if (boundaryFlag != 0) {
        if (pVertexFlag[n] == 0) do_flag = 0;
      }

      if (do_flag)
        SetInitialSingle<CL>(n, pVc, p, pVertexPotential,
                             state, G, time, Px, Py,
                             denspow, cs0, cspow);
    }

  }

  try {
    // Add KH eigenvector
    if (p == PROBLEM_KH)
      KHAddEigenVector();
    if (p == PROBLEM_SOURCE && CL == CL_CART_EULER)
      RTAddEigenVector();
  }
  catch (...) {
    std::cout << "Warning: reading eigenvector file failed!" << std::endl;
    std::cout << "Running simulation without adding eigenvector!"
              << std::endl;
  }
}

//##############################################################################
// Instantiate
//##############################################################################

template void Simulation<real, CL_ADVECT>::SetInitial(real time,
                                                      int boundaryFlag);
template void Simulation<real, CL_BURGERS>::SetInitial(real time,
                                                       int boundaryFlag);
template void Simulation<real3, CL_CART_ISO>::SetInitial(real time,
                                                         int boundaryFlag);
template void Simulation<real3, CL_CYL_ISO>::SetInitial(real time,
                                                        int boundaryFlag);
template void Simulation<real4, CL_CART_EULER>::SetInitial(real time,
                                                           int boundaryFlag);

}  // namespace astrix
