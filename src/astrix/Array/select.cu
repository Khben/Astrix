#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file select.cu
\brief Functions for selecting part of Array

\section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <iostream>

#include "./array.h"
#include "../Common/cudaLow.h"

namespace astrix {

//###################################################
//
//###################################################

template<class T>
__global__ void
devSelectLargerThan(int N, T *data, T value, int *pSelectFlag)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < N) {
    int ret = 0;
    if (data[i] > value) ret = 1;
    pSelectFlag[i] = ret;

    i += gridDim.x*blockDim.x;
  }
}

//###################################################
//
//###################################################

template <class T>
template <class S>
int Array<T>::SelectLargerThan(T value, Array<S> *A)
{
  Array<int> *selectFlag = new Array<int>(1, cudaFlag, size);
  int *pSelectFlag = selectFlag->GetPointer();
  Array<int> *selectFlagScan = new Array<int>(1, cudaFlag, size);

  if (cudaFlag == 1) {
    int nThreads = 128;
    int nBlocks = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devSelectLargerThan<T>,
                                       (size_t) 0, 0);

    devSelectLargerThan<<<nBlocks, nThreads>>>(size, deviceVec, value,
                                               pSelectFlag);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
  } else {
    for (unsigned int i = 0; i < size; i++) {
      int ret = 0;
      if (hostVec[i] > value) ret = 1;
      pSelectFlag[i] = ret;
    }
  }

  int nSelect = selectFlag->ExclusiveScan(selectFlagScan);

  Compact(nSelect, selectFlag, selectFlagScan);
  A->Compact(nSelect, selectFlag, selectFlagScan);

  delete selectFlag;
  delete selectFlagScan;

  return nSelect;
}

//###################################################
//
//###################################################

template<class T>
__global__ void
devSelectWhereDifferent(int N, T *data, T *compareData, int *pSelectFlag)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < N) {
    int ret = 0;
    if (data[i] != compareData[i]) ret = 1;
    pSelectFlag[i] = ret;

    i += gridDim.x*blockDim.x;
  }
}

//###################################################
//
//###################################################

template <class T>
template <class S>
int Array<T>::SelectWhereDifferent(Array<T> *A, Array<S> *B)
{
  Array<int> *selectFlag = new Array<int>(1, cudaFlag, size);
  int *pSelectFlag = selectFlag->GetPointer();
  Array<int> *selectFlagScan = new Array<int>(1, cudaFlag, size);

  T *compareData = A->GetPointer();

  if (cudaFlag == 1) {
    int nThreads = 128;
    int nBlocks = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devSelectWhereDifferent<T>,
                                       (size_t) 0, 0);

    devSelectWhereDifferent<<<nBlocks, nThreads>>>(size, deviceVec,
                                                   compareData, pSelectFlag);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
  } else {
    for (unsigned int i = 0; i < size; i++) {
      int ret = 0;
      if (hostVec[i] != compareData[i]) ret = 1;
      pSelectFlag[i] = ret;
    }
  }

  int nSelect = selectFlag->ExclusiveScan(selectFlagScan);

  Compact(nSelect, selectFlag, selectFlagScan);
  B->Compact(nSelect, selectFlag, selectFlagScan);

  delete selectFlag;
  delete selectFlagScan;

  return nSelect;
}

//###################################################
// Instantiate
//###################################################

template int Array<int>::SelectLargerThan(int value, Array<double> *A);
template int Array<int>::SelectLargerThan(int value, Array<double2> *A);
template int Array<int>::SelectLargerThan(int value, Array<float> *A);
template int Array<int>::SelectLargerThan(int value, Array<float2> *A);

template int Array<int>::SelectWhereDifferent(Array<int> *A,
                                              Array<double> *B);
template int Array<int>::SelectWhereDifferent(Array<int> *A,
                                              Array<double2> *B);
template int Array<int>::SelectWhereDifferent(Array<int> *A,
                                              Array<float> *B);
template int Array<int>::SelectWhereDifferent(Array<int> *A,
                                              Array<float2> *B);

}
