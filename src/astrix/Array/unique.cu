#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file unique.cu
\brief Functions for compacting array

*/ /* \section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "./array.h"
#include "../Common/cudaLow.h"

namespace astrix {

//######################################################################
//! Kernel: At a unique entry of A, i, (ignoring ignoreValue) set hostVec[B[i]] = value.
//######################################################################

template<class T>
__global__ void
devScatterUnique(int N, int *pA, int *pB, T* srcArray, int ignoreValue, T value)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + 1;

  while (i < N) {
    if (pA[i - 1] == pA[i] && pA[i] != ignoreValue)
        srcArray[pB[i]] = value;

    i += gridDim.x*blockDim.x;
  }
}

//###################################################
// At a unique entry of A, i, (ignoring ignoreValue)
// set hostVec[B[i]] = value.
//###################################################

template <class T>
void Array<T>::ScatterUnique(Array<int> *A, Array<int> *B,
                             int maxIndex, int ignoreValue, T value)
{
  int *pA = A->GetPointer();
  int *pB = B->GetPointer();

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize
      (&nBlocks, &nThreads, devScatterUnique<T>, (size_t) 0, 0);

    devScatterUnique<<<nBlocks, nThreads>>>
      (maxIndex, pA, pB, deviceVec, ignoreValue, value);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int i = 1; i < maxIndex; i++)
      if (pA[i - 1] == pA[i] && pA[i] != ignoreValue)
        hostVec[pB[i]] = value;
  }
}

//###################################################

template void Array<int>::ScatterUnique(Array<int> *A, Array<int> *B,
                                        int maxIndex, int ignoreValue,
                                        int value);
template void Array<float2>::ScatterUnique(Array<int> *A, Array<int> *B,
                                           int maxIndex, int ignoreValue,
                                           float2 value);

}  // namespace astrix
