#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file compact.cu
\brief Functions for compacting array

\section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <thrust/remove.h>
#include <thrust/distance.h>
#include <thrust/device_vector.h>

#include "./array.h"
#include "../Common/cudaLow.h"

namespace astrix {

//###################################################
// Remove values from array
//###################################################

template <class T>
int Array<T>::RemoveValue(T value)
{
  int newSize = 0;

  if (cudaFlag == 1) {
    thrust::device_ptr<T> dev_ptr(deviceVec);
    thrust::device_ptr<T> iter;

    iter = thrust::remove(dev_ptr, dev_ptr + size, value);

    newSize = iter - dev_ptr;
  }
  if (cudaFlag == 0) {
    T *iter = thrust::remove(hostVec, hostVec + size, value);
    newSize = iter - hostVec;
  }

  return newSize;
}

//###################################################
// Remove values from array
//###################################################

template <class T>
int Array<T>::RemoveValue(T value, int maxIndex)
{
  int newSize = 0;

  if (cudaFlag == 1) {
    thrust::device_ptr<T> dev_ptr(deviceVec);
    thrust::device_ptr<T> iter;

    iter = thrust::remove(dev_ptr, dev_ptr + maxIndex, value);

    newSize = iter - dev_ptr;
  }
  if (cudaFlag == 0) {
    int *iter = thrust::remove(hostVec, hostVec + maxIndex, value);
    newSize = iter - hostVec;
  }

  return newSize;
}

//######################################################################
//! Kernel for compacting array
//######################################################################

template<class T>
__global__ void
devCompact(int N, T *destArray, T *srcArray,
           int *keepFlag, int *keepFlagScan,
           int realSize, int nDims)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < N) {
    if (keepFlag[i] == 1) {
      for (unsigned int n = 0; n < nDims; n++)
        destArray[keepFlagScan[i] + n*realSize] = srcArray[i + n*realSize];
    }
    i += gridDim.x*blockDim.x;
  }
}

//###################################################
// Compact array
//###################################################

template <class T>
void Array<T>::Compact(int nKeep,
                       Array<int> *keepFlag,
                       Array<int> *keepFlagScan)
{
  int *pKeepFlag = keepFlag->GetPointer();
  int *pKeepFlagScan = keepFlagScan->GetPointer();

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devCompact<T>,
                                       (size_t) 0, 0);

    T *temp;
    gpuErrchk(hipMalloc(reinterpret_cast<void**>(&temp),
                         nDims*realSize*sizeof(T)));
    gpuErrchk(hipMemcpy(temp, deviceVec,
                         nDims*realSize*sizeof(T),
                         hipMemcpyDeviceToDevice));

    devCompact<<<nBlocks, nThreads>>>(size, deviceVec, temp,
                                      pKeepFlag, pKeepFlagScan,
                                      realSize, nDims);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk(hipFree(temp));
  }

  if (cudaFlag == 0) {
    // Temporary array
    T *temp = (T *)malloc(nDims*realSize*sizeof(T));
    memcpy(temp, hostVec, nDims*realSize*sizeof(T));

    for (unsigned int n = 0; n < nDims; n++)
      for (unsigned int i = 0; i < size; i++)
        if (pKeepFlag[i] == 1)
          hostVec[pKeepFlagScan[i] + n*realSize] = temp[i + n*realSize];

    free(temp);
  }

  SetSize(nKeep);
}

//###################################################
// Instantiate
//###################################################

template void Array<float>::Compact(int nKeep,
                                    Array<int> *keepFlag,
                                    Array<int> *keepFlagScan);

//###################################################

template void Array<double>::Compact(int nKeep,
                                     Array<int> *keepFlag,
                                     Array<int> *keepFlagScan);

//###################################################

template void Array<int>::Compact(int nKeep,
                                  Array<int> *keepFlag,
                                  Array<int> *keepFlagScan);

//###################################################

template void
Array<unsigned int>::Compact(int nKeep,
                             Array<int> *keepFlag,
                             Array<int> *keepFlagScan);

//###################################################

template void Array<int2>::Compact(int nKeep,
                                   Array<int> *keepFlag,
                                   Array<int> *keepFlagScan);

template void Array<int3>::Compact(int nKeep,
                                   Array<int> *keepFlag,
                                   Array<int> *keepFlagScan);
template void Array<float2>::Compact(int nKeep,
                                     Array<int> *keepFlag,
                                     Array<int> *keepFlagScan);
template void Array<float3>::Compact(int nKeep,
                                     Array<int> *keepFlag,
                                     Array<int> *keepFlagScan);
template void Array<float4>::Compact(int nKeep,
                                     Array<int> *keepFlag,
                                     Array<int> *keepFlagScan);
template void Array<double2>::Compact(int nKeep,
                                     Array<int> *keepFlag,
                                     Array<int> *keepFlagScan);
template void Array<double3>::Compact(int nKeep,
                                     Array<int> *keepFlag,
                                     Array<int> *keepFlagScan);
template void Array<double4>::Compact(int nKeep,
                                     Array<int> *keepFlag,
                                     Array<int> *keepFlagScan);


template int Array<int>::RemoveValue(int value);
template int Array<int>::RemoveValue(int value, int maxIndex);
template int Array<unsigned int>::RemoveValue(unsigned int value);

}  // namespace astrix
