#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file findtriangles.cu
\brief File containing function to find triangles containing point (x,y).*/

#include <iostream>
#include <stdexcept>

#include "../../Common/definitions.h"
#include "../../Array/array.h"
#include "../Predicates/predicates.h"
#include "./refine.h"
#include "../triangleLow.h"
#include "../../Common/cudaLow.h"
#include "../../Common/nvtxEvent.h"
#include "../Connectivity/connectivity.h"
#include "../Param/meshparameter.h"
#include "../../Common/profile.h"

namespace astrix {

//#########################################################################
/*! \brief Find triangle to put (x,y) in

Walk through the mesh, starting at \a tStart, trying to find a triangle containing point (x, y). Returns the index of the triangle that the point (x, y) is located in. If the point is located exactly on an edge, the index of this edge is stored in \a edgeIndex and the function will return -1. 
 
\param tStart Triangle to start in
\param x X-coordinate of point to find triangle for
\param y Y-coordinate of point to find triangle for
\param nTriangle Total number of triangles in Mesh
\param edgeIndex If point lies exactly on edge, the index of the edge will be stored here
\param *pVertX Pointer to x-coordinates of vertices
\param *pVertY Pointer to y-coordinates of vertices
\param *tv1 Pointer to first vertex of triangle 
\param *tv2 Pointer to second vertex of triangle 
\param *tv3 Pointer to third vertex of triangle 
\param *te1 Pointer to first edge of triangle 
\param *te2 Pointer to second edge of triangle 
\param *te3 Pointer to third edge of triangle 
\param *et1 Pointer to first triangle neighbouring edge
\param *et2 Pointer to second triangle neighbouring edge
\param *pred Pointer to initialised Predicates object
\param *pParam Pointer to initialised Predicates parameter vector
\param nVertex Total number of vertices in Mesh
\param Px Periodic domain size x
\param Py Periodic domain size y
\param printFlag Flag whether to print triangle walk on screen (for debugging purposes)*/
//#########################################################################

__host__ __device__
int FindTriangle(int tStart, real& x, real& y, int nTriangle,
		 const real2* __restrict__ pVc,
		 const int3* __restrict__ pTv,
		 const int3* __restrict__ pTe,
		 const int2* __restrict__ pEt,
		 const Predicates *pred, real *pParam,
		 int nVertex, real Px, real Py,
		 int printFlag, int& nSteps)
{
  const real zero = (real) 0.0;
  
  int ret = -1;

  // Adjust starting triangle if necessary
  if (tStart < 0 || tStart >= nTriangle) tStart = 0;
  int t = tStart;
  int tPrev = -1;
  int finished = 0;
  int crossTimes = 0;
  // Vertices belonging to edge we are moving across
  int edgeV2 = -1;// = pTv[t].y;
  int edgeCrossed = -1;
  int edgeIndex = -1;
  nSteps = 0;
 
  while (finished == 0 && nSteps <= 2*nTriangle) {
    int tNext = t;

    int a = pTv[t].x;
    int b = pTv[t].y;
    int c = pTv[t].z;

    //if (nSteps == 0) edgeV2 = b;
    
    real ax, bx, cx, ay, by, cy;
    GetTriangleCoordinates(pVc, a, b, c,
			   nVertex, Px, Py,
			   ax, bx, cx, ay, by, cy);

    int e1 = pTe[t].x;
    int e2 = pTe[t].y;
    int e3 = pTe[t].z;

    // PERIODIC
    // Translate (x, y) such that edge (V1, V2) is part of t
    if (edgeCrossed >= 0) {
      int v = -1;
      if (edgeCrossed == e1) v = a;
      if (edgeCrossed == e2) v = b;
      if (edgeCrossed == e3) v = c;
      
      TranslateVertexToVertex(v, edgeV2, Px, Py, nVertex, x, y);
    }
 
    
    real dx = x, dy = y;

    //real A1 = pred->orient2d(dx, dy, ax, ay, bx, by, pParam);
    //real A2 = pred->orient2d(dx, dy, bx, by, cx, cy, pParam);
    //real A3 = pred->orient2d(dx, dy, cx, cy, ax, ay, pParam);

    real detleft = (dx - bx) * (ay - by);
    real detright = (dy - by) * (ax - bx);
    real A1 = detleft - detright;
    detleft = (dx - cx) * (by - cy);
    detright = (dy - cy) * (bx - cx);
    real A2 = detleft - detright;
    detleft = (dx - ax) * (cy - ay);
    detright = (dy - ay) * (cx - ax);
    real A3 = detleft - detright;

    edgeCrossed = -1;
    
    if (A1 < zero) {
      edgeCrossed = e1;
      edgeV2 = b;
    }
    if (A2 < zero && A2 < A1) {
      edgeCrossed = e2;
      edgeV2 = c;
    }
    if (A3 < zero && A3 < A1 && A3 < A2) {
      edgeCrossed = e3;
      edgeV2 = a;
    }

    if (edgeCrossed != -1) {
      int t1 = pEt[edgeCrossed].x;
      int t2 = pEt[edgeCrossed].y;
      tNext = t1;
      if(tNext == t) tNext = t2;
    }
    
    // Keep track how many times we go back to triangle we came from
    if (tNext == tPrev) crossTimes++; else crossTimes = 0;
    
    // Not moving to new triangle: A1, A2, A3 >= 0
    if (tNext == t) {
      // Must have found edge if any of A's are zero
      if (A1 == zero) edgeIndex = e1;
      if (A2 == zero) edgeIndex = e2;
      if (A3 == zero) edgeIndex = e3;

      // Otherwise, we have found a triangle
      finished = 1;     
    }

    // Either moved across segment or back to previous triangle for 2nd time
    if (tNext == -1 || crossTimes > 1) {
      // Choose best edge (minimum A)
      if (A1 < A2 && A1 < A3) edgeIndex = e1;
      if (A2 < A1 && A2 < A3) edgeIndex = e2;
      if (A3 < A2 && A3 < A1) edgeIndex = e3;
      
      finished = 1;
    }

    tPrev = t;
    t = tNext;
    
    nSteps++;
  }

  // Triangle found
  if (finished == 1) 
    if (edgeIndex == -1) ret = t; else ret = edgeIndex + nTriangle;
  
  return ret;
}
    
//######################################################################
/*! \brief Kernel finding triangles to put \a nRefine points in

Walk through the mesh trying to find triangles containing points specified in \a refineX and \a refineY.   
 
\param nRefine Total number of points to find triangles for
\param nTriangle Total number of triangles in Mesh
\param *refineIndex Pointer to triangles spawning the points to be inserted. These triangles will serve as starting locations for the walk through the Mesh
\param *pTriangleAdd Pointer to output array containing triangles found, or -1 if no triangle could be found
\param *pEdgeAdd Pointer to output array containing edges found, or -1 if no edge could be found 
\param refineX X-coordinates of points to find triangles for
\param refineY Y-coordinates of points to find triangles for
\param *pVertX Pointer to x-coordinates of vertices
\param *pVertY Pointer to y-coordinates of vertices
\param *tv1 Pointer to first vertex of triangle 
\param *tv2 Pointer to second vertex of triangle 
\param *tv3 Pointer to third vertex of triangle 
\param *te1 Pointer to first edge of triangle 
\param *te2 Pointer to second edge of triangle 
\param *te3 Pointer to third edge of triangle 
\param *et1 Pointer to first triangle neighbouring edge
\param *et2 Pointer to second triangle neighbouring edge
\param *pred Pointer to initialised Predicates object
\param *pParam Pointer to initialised Predicates parameter vector
\param nVertex Total number of vertices in Mesh
\param Px Periodic domain size x
\param Py Periodic domain size y*/
//######################################################################

__global__ void 
devFindTriangles(int nRefine, int nTriangle, 
		 int *refineIndex, int *pElementAdd, real2 *pVcAdd,
		 const real2* __restrict__ pVc,
		 const int3* __restrict__ pTv,
		 const int3* __restrict__ pTe,
		 const int2* __restrict__ pEt,
		 const Predicates *pred, real *pParam,
		 int nVertex, real Px, real Py)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nRefine) {
    // Coordinates of point to be inserted
    real x = pVcAdd[i].x;
    real y = pVcAdd[i].y;
      
    int a = 0;
    
    // Find triangle or edge to place vertex
    int t = FindTriangle(refineIndex[i], x, y, nTriangle,
			 pVc, pTv, pTe, pEt,
			 pred, pParam,
			 nVertex, Px, Py, 0, a);

    pVcAdd[i].x = x;
    pVcAdd[i].y = y;
   
    pElementAdd[i] = t;
    
    i += gridDim.x*blockDim.x;
  }
}
  
//#########################################################################
/*! When we have created a list of points (x,y) to insert into the mesh in \a vertexCoordinatesAdd, we have to find triangles to put these points in. This is done by walking through the grid, starting from the triangle that initiated the point, until we have found either a suitable triangle or a suitable edge. Results are put in \a elementAdd

\param *connectivity Pointer to basic Mesh data
\param *meshParameter Pointer to mesh parameters
\param *predicates Exact geomentric predicates*/  
//#########################################################################

void Refine::FindTriangles(Connectivity * const connectivity,
			   const MeshParameter *meshParameter,
			   const Predicates *predicates)
{
#ifdef TIME_ASTRIX
  hipEvent_t start, stop;
  float elapsedTime = 0.0f;
  hipEventCreate(&start);
  hipEventCreate(&stop);
#endif

  nvtxEvent *nvtxFind = new nvtxEvent("FindTriangles", 1);

  int nVertex = connectivity->vertexCoordinates->GetSize();
  int nTriangle = connectivity->triangleVertices->GetSize();
  int nRefine = elementAdd->GetSize();
  
  real2 *pVc = connectivity->vertexCoordinates->GetPointer();
  int3 *pTv = connectivity->triangleVertices->GetPointer();
  int3 *pTe = connectivity->triangleEdges->GetPointer();
  int2 *pEt = connectivity->edgeTriangles->GetPointer();

  int *pBadTriangles = badTriangles->GetPointer();
  int *pElementAdd = elementAdd->GetPointer();
  real2 *pVcAdd = vertexCoordinatesAdd->GetPointer();
   
  real *pParam = predicates->GetParamPointer(cudaFlag);

  real Px = meshParameter->maxx - meshParameter->minx;
  real Py = meshParameter->maxy - meshParameter->miny;

  // Find trangles to put new vertices in
  if (cudaFlag == 1) {
    int nBlocks = 26;
    int nThreads = 512;
    
    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
    				       devFindTriangles, 
    				       (size_t) 0, 0);

#ifdef TIME_ASTRIX
    hipEventRecord(start, 0);
#endif
    devFindTriangles<<<nBlocks, nThreads>>>
      (nRefine, nTriangle, pBadTriangles,
       pElementAdd, pVcAdd,
       pVc, pTv, pTe, pEt,
       predicates, pParam,
       nVertex, Px, Py);
#ifdef TIME_ASTRIX
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
#endif      
    
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());      
  } else {
#ifdef TIME_ASTRIX
    hipEventRecord(start, 0);
#endif
    for (int i = 0; i < nRefine; i++) {
      // Coordinates of point to be inserted
      real x = pVcAdd[i].x;
      real y = pVcAdd[i].y;

      int printFlag = 0;
      
      int nSteps = 0;

      // Find triangle or edge to place vertex
      int t = FindTriangle(pBadTriangles[i], x, y, nTriangle,
			   pVc, pTv, pTe, pEt,
			   predicates, pParam,
			   nVertex, Px, Py, printFlag,
			   nSteps);

      pVcAdd[i].x = x;
      pVcAdd[i].y = y;
      
      pElementAdd[i] = t;

      if (t == -1) {
	std::cout << std::endl 
		  << "Error in FindTriangles: no triangle or edge found!"
		  << std::endl
		  << "Vertex location: " << x << " " << y << std::endl
		  << "Starting triangle: " << pBadTriangles[i] << std::endl;
	throw std::runtime_error("");
	//int qq; std::cin >> qq;
      }
    }
#ifdef TIME_ASTRIX
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
#endif      

  }

#ifdef TIME_ASTRIX
  hipEventElapsedTime(&elapsedTime, start, stop);
  WriteProfileFile("FindTriangle.prof", nRefine, elapsedTime, cudaFlag);
#endif

  delete nvtxFind;
}

}
