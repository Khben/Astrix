#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file flagsegment.cu
\brief Functions for determining how many and which points to be added will be placed on segments*/
#include <iostream>

#include "../../Common/definitions.h"
#include "../../Array/array.h"
#include "./refine.h"
#include "../../Common/cudaLow.h"
#include "../Connectivity/connectivity.h"

namespace astrix {

//######################################################################
/*! \brief Flag whether vertex \i will be placed on segment

\param i Index of point to be added
\param *pEdgeAdd Pointer to array containing edges onto which to place points
\param *et1 Pointer to first triangle neighbouring edge
\param *et2 Pointer to second triangle neighbouring edge
\param *pOnSegmentFlag Pointer to output array, will be 1 if \i will be placed on segment, 0 otherwise*/
//######################################################################

__host__ __device__
void FlagOnSegmentSingle(int i, int *pElementAdd, int2 *pEt,
			 unsigned int *pOnSegmentFlag, int nTriangle)
{
  int e = pElementAdd[i] - nTriangle;
  unsigned int ret = 0;
  if (e >= 0) 
    ret = pEt[e].x == -1 || pEt[e].y == -1;
  pOnSegmentFlag[i] = ret;
}
    
//######################################################################
/*! \brief Kernel flagging whether vertices will be placed on segment

\param nRefine Number of points to be added
\param *pEdgeAdd Pointer to array containing edges onto which to place points
\param *et1 Pointer to first triangle neighbouring edge
\param *et2 Pointer to second triangle neighbouring edge
\param *pOnSegmentFlag Pointer to output array, will be 1 if \i will be placed on segment, 0 otherwise*/
//######################################################################

__global__ void
devFlagOnSegment(int nRefine, int *pElementAdd, int2 *pEt,
		 unsigned int *pOnSegmentFlag, int nTriangle)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nRefine) {
    FlagOnSegmentSingle(i, pElementAdd, pEt, pOnSegmentFlag, nTriangle);
  
    i += blockDim.x*gridDim.x;
  }
}
  
//######################################################################
/*! Find which of the points to be added will be placed on segments. This is not difficult using the \a edgeAdd Array, which contains the indices of edges to place points on. All we have to do is find out which of these edges are in fact segments. Returns total number of points to be placed on segments.

\param *onSegmentFlagScan Pointer to output array: a scanned version of the Array containing flags*/
//######################################################################

int Refine::FlagSegment(Connectivity * const connectivity,
			Array<unsigned int> * const onSegmentFlagScan)
{
  int nRefine = elementAdd->GetSize();
  int nTriangle = connectivity->triangleVertices->GetSize();
  int *pElementAdd = elementAdd->GetPointer();
  
  Array<unsigned int> *onSegmentFlag = 
    new Array<unsigned int>(1, cudaFlag, (unsigned int) nRefine);

  // Flag points to be inserted on segment
  unsigned int *pOnSegmentFlag = onSegmentFlag->GetPointer();
  int2 *pEt = connectivity->edgeTriangles->GetPointer();
  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;
    
    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
				       devFlagOnSegment, 
				       (size_t) 0, 0);

    devFlagOnSegment<<<nBlocks, nThreads>>>
      (nRefine, pElementAdd, pEt, pOnSegmentFlag, nTriangle);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
  } else {
    for (int i = 0; i < nRefine; i++) 
      FlagOnSegmentSingle(i, pElementAdd, pEt, pOnSegmentFlag, nTriangle);
  }
  
  onSegmentFlagScan->SetSize(nRefine);

  int nOnSegment = onSegmentFlag->ExclusiveScan(onSegmentFlagScan);

  delete onSegmentFlag;

  return nOnSegment;
}

}
