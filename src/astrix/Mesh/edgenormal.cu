#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file normalarea.cu
\brief Functions for ccalculating triangle normals and triangle and vertex areas*/
#include <iostream>

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "./mesh.h"
#include "./triangleLow.h"
#include "../Common/cudaLow.h"
#include "../Common/inlineMath.h"
#include "./Connectivity/connectivity.h"
#include "./Param/meshparameter.h"

namespace astrix {

//######################################################################
/*! \brief Calculate normals and edge lengths for triangle \a n

\param n Index of triangle to consider
\param nTriangle Total number of triangles in Mesh
\param *tv1 Pointer to first vertex of triangle 
\param *tv2 Pointer to second vertex of triangle 
\param *tv3 Pointer to third vertex of triangle 
\param *pVertX Pointer to x-coordinates of vertices
\param *pVertY Pointer to y-coordinates of vertices
\param *triNx Pointer to x-components of triangle normals (output)
\param *triNy Pointer to y-components of triangle normals (output)
\param *triL Pointer to array of triangle edge lengths (output)
\param nVertex Total number of vertices in Mesh
\param Px Periodic domain size x
\param Py Periodic domain size y*/
//######################################################################

__host__ __device__
void CalcNormalEdgeSingle(int n, int nTriangle, int3 *pTv,
			  real2 *pVc,
			  //real *triNx, real *triNy,
			  real2 *pTn1, real2 *pTn2, real2 *pTn3, 
			  real3 *triL, int nVertex,
			  real Px, real Py)
{
  const real zero  = (real) 0.0;
  const real half  = (real) 0.5;

  int a = pTv[n].x;
  int b = pTv[n].y;
  int c = pTv[n].z;
  
  real ax, bx, cx, ay, by, cy;
  GetTriangleCoordinates(pVc, a, b, c,
			 nVertex, Px, Py,
			 ax, bx, cx, ay, by, cy);

  // Vector along face
  real facedx = bx - cx;
  real facedy = by - cy;
    
  // Vector orthogonal to face
  real nx = facedy;
  real ny = -facedx;
    
  // Check if pointing inward
  real innerprod = nx*ax + ny*ay - half*(nx*bx + ny*by + nx*cx + ny*cy);
  
  // If pointing outward, reverse
  if(innerprod < zero){
    nx = -nx;
    ny = -ny;
  }
  
  // Scale to length unity
  real inverselength = pow(nx*nx + ny*ny, -half);
  
  //triNx[0*nTriangle+n] = nx*inverselength;
  //triNy[0*nTriangle+n] = ny*inverselength;      
  pTn1[n].x = nx*inverselength;
  pTn1[n].y = ny*inverselength;      
  
  // Vector along face
  facedx = cx - ax;
  facedy = cy - ay;
  
  // Vector orthogonal to face
  nx = facedy;
  ny = -facedx;
  
  // Check if pointing inward
  innerprod = nx*bx + ny*by - half*(nx*cx + ny*cy + nx*ax + ny*ay);
  
  // If pointing outward, reverse
  if(innerprod < zero){
    nx = -nx;
    ny = -ny;
  }
  
  // Scale to length unity
  inverselength = pow(nx*nx + ny*ny, -half);
  
  pTn2[n].x = nx*inverselength;
  pTn2[n].y = ny*inverselength;
  
  // Vector along face
  facedx = ax - bx;
  facedy = ay - by;
  
  // Vector orthogonal to face
  nx = facedy;
  ny = -facedx;
  
  // Check if pointing inward
  innerprod = nx*cx + ny*cy - half*(nx*ax + ny*ay + nx*bx + ny*by);
  
  // If pointing outward, reverse
  if(innerprod < zero){
    nx = -nx;
    ny = -ny;
  }
  
  // Scale to length unity
  inverselength = pow(nx*nx + ny*ny, -half);

  pTn3[n].x = nx*inverselength;
  pTn3[n].y = ny*inverselength;      
    
  // l[n][i] = length of face of triangle n opposite to vertex i
  //triL[0*nTriangle+n] = sqrt(Sq(bx - cx) + Sq(by - cy));
  //triL[1*nTriangle+n] = sqrt(Sq(ax - cx) + Sq(ay - cy));
  //triL[2*nTriangle+n] = sqrt(Sq(bx - ax) + Sq(by - ay));
  triL[n].x = sqrt(Sq(bx - cx) + Sq(by - cy));
  triL[n].y = sqrt(Sq(ax - cx) + Sq(ay - cy));
  triL[n].z = sqrt(Sq(bx - ax) + Sq(by - ay));
}
  
//######################################################################
/*! \brief Kernel calculating normals and edge lengths for all triangles

\param nTriangle Total number of triangles in Mesh
\param *tv1 Pointer to first vertex of triangle 
\param *tv2 Pointer to second vertex of triangle 
\param *tv3 Pointer to third vertex of triangle 
\param *pVertX Pointer to x-coordinates of vertices
\param *pVertY Pointer to y-coordinates of vertices
\param *triNx Pointer to x-components of triangle normals (output)
\param *triNy Pointer to y-components of triangle normals (output)
\param *triL Pointer to array of triangle edge lengths (output)
\param nVertex Total number of vertices in Mesh
\param Px Periodic domain size x
\param Py Periodic domain size y*/ 
//######################################################################

__global__ void 
devCalcNormalEdge(int nTriangle, int3 *pTv, real2 *pVc,
		  //real *triNx, real *triNy,
		  real2 *pTn1, real2 *pTn2, real2 *pTn3, 
		  real3 *triL, int nVertex,
		  real Px, real Py)
{
  // n = triangle number
  int n = blockIdx.x*blockDim.x + threadIdx.x; 

  while(n < nTriangle){
    CalcNormalEdgeSingle(n, nTriangle, pTv,
			 pVc,
			 //triNx, triNy,
			 pTn1, pTn2, pTn3,
			 triL,
			 nVertex, Px, Py);

    n += blockDim.x*gridDim.x;
  }
}
  
//######################################################################
/*! Calculate inward-pointing normals (length unity) and edge lengths for all triangles in Mesh*/
//######################################################################

void Mesh::CalcNormalEdge()
{
  real2 *pVc = connectivity->vertexCoordinates->GetPointer();
  int3 *pTv = connectivity->triangleVertices->GetPointer();

  int nTriangle = connectivity->triangleVertices->GetSize();
  int nVertex = connectivity->vertexCoordinates->GetSize();

  triangleEdgeNormals->SetSize(nTriangle);
  real2 *pTn1 = triangleEdgeNormals->GetPointer(0);
  real2 *pTn2 = triangleEdgeNormals->GetPointer(1);
  real2 *pTn3 = triangleEdgeNormals->GetPointer(2);
  
  triangleEdgeLength->SetSize(nTriangle);
  real3 *triL = triangleEdgeLength->GetPointer();

  real Px = meshParameter->maxx - meshParameter->minx;
  real Py = meshParameter->maxy - meshParameter->miny;

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
				       devCalcNormalEdge, 
				       (size_t) 0, 0);

    devCalcNormalEdge<<<nBlocks, nThreads>>>
      (nTriangle, pTv, pVc, pTn1, pTn2, pTn3, triL, nVertex, Px, Py);
    
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int n = 0; n < nTriangle; n++) 
      CalcNormalEdgeSingle(n, nTriangle, pTv, pVc,
			   pTn1, pTn2, pTn3, triL,
			   nVertex, Px, Py);
  }
}

}
