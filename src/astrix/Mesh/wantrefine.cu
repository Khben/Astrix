#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file refine_wantrefine.cu
\brief File containing functions to determine which triangles need refining based on an estimate of the local truncation error.*/
#include <iostream>

#include "../Common/definitions.h"
#include "../Array/array.h"
#include "./mesh.h"
#include "../Common/cudaLow.h"
#include "./Param/meshparameter.h"

namespace astrix {

// #########################################################################
/*! \brief Check if triangle i needs refining based on ErrorEstimate

\param i Index of triangle to consider
\param *pErrorEstimate Pointer to array with estimates of local truncation error
 (LTE)
\param maxError Limit of LTE above which to flag triangle for refinement
\param minError Limit of LTE below which to flag triangle for refinement
\param *pWantRefine Pointer to output array: 1 if triangle needs refining, -1 if it can be coarsened, 0 if nothing needs to happen*/
// #########################################################################

__host__ __device__
void FillWantRefineSingle(int i, real *pErrorEstimate,
			  real maxError, real minError,
			  int *pWantRefine)
{
  int ret = 0;
  
  if (pErrorEstimate[i] > maxError) ret = 1;
  if (pErrorEstimate[i] < minError) ret = -1;
  
  pWantRefine[i] = ret; 
}

//######################################################################
/*! \brief Kernel checking if triangles need refining based on ErrorEstimate

\param nTriangle Total number of triangles in Mesh
\param *pErrorEstimate Pointer to array with estimates of local truncation error
 (LTE)
\param maxError Limit of LTE above which to flag triangle for refinement
\param minError Limit of LTE below which to flag triangle for refinement
\param *pWantRefine Pointer to output array: 1 if triangle needs refining, -1 if it can be coarsened, 0 if nothing needs to happen*/
//######################################################################

__global__ void
devFillWantRefine(int nTriangle, real *pErrorEstimate,
		  real maxError, real minError, int *pWantRefine)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nTriangle) {
    FillWantRefineSingle(i, pErrorEstimate, maxError, minError, pWantRefine);

    i += blockDim.x*gridDim.x;
  }
}

// #########################################################################
/*! Flag triangles for refinement or coarsening based on an estimate of the local truncation error (LTE). First the LTE is computed; then we fill the Array triangleWantRefine with either 1 (triangle needs refining), -1 (triangle can be coarsened) or 0 (nothing needs to happen).

\param *vertexState Pointer to Array containing state vector (density etc). Needed to compute LTE
\param specificHeatRatio Ratio of specific heats*/ 
// #########################################################################

void Mesh::FillWantRefine(Array<realNeq> *vertexState, real specificHeatRatio)
{
  CalcErrorEstimate(vertexState, specificHeatRatio);
  real *pErrorEstimate = triangleErrorEstimate->GetPointer();
  int *pWantRefine = triangleWantRefine->GetPointer();

  real minError = meshParameter->minError;
  real maxError = meshParameter->maxError;
  
  if (cudaFlag == 1) {    
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
				       devFillWantRefine, 
				       (size_t) 0, 0);

    devFillWantRefine<<<nBlocks, nThreads>>>
      (nTriangle, pErrorEstimate, maxError, minError, pWantRefine);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int i = 0; i < nTriangle; i++) 
      FillWantRefineSingle(i, pErrorEstimate, maxError, minError, pWantRefine);
  }
}

}
