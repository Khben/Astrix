#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file lock.cu
\brief Functions for locking vertices surrounding deletion point to decide which can be deleted in parallel

\section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/
#include <iostream>

#include "../../Common/definitions.h"
#include "../../Array/array.h"
#include "./coarsen.h"
#include "../../Common/cudaLow.h"
#include "../Connectivity/connectivity.h"
#include "../../Common/atomic.h"

namespace astrix {

//#########################################################################
//#########################################################################

__host__ __device__
void LockTrianglesSingle(int vRemove, int tStart,
                         int3 *pTv, int3 *pTe, int2 *pEt,
                         int nVertex, int tTarget,
                         int randomInt,
                         int *pTriangleLock)
{
  int t = tStart;
  int tPrev = -1;
  int finished = 0;
  while (!finished) {
    // Edge to cross to next triangle
    int eCross = -1;

    int a = pTv[t].x;
    int b = pTv[t].y;
    int c = pTv[t].z;

    while (a >= nVertex) a -= nVertex;
    while (b >= nVertex) b -= nVertex;
    while (c >= nVertex) c -= nVertex;
    while (a < 0) a += nVertex;
    while (b < 0) b += nVertex;
    while (c < 0) c += nVertex;

    int3 E = pTe[t];
    // Edge on boundary of 'cavity'
    int eBound = -1;
    if (a == vRemove) {
      eCross = E.x;
      eBound = E.y;
    }
    if (b == vRemove) {
      eCross = E.y;
      eBound = E.z;
    }
    if (c == vRemove) {
      eCross = E.z;
      eBound = E.x;
    }

    // Set pTiC to maximum of pTiC and pRandom
    int old = AtomicMax(&(pTriangleLock[t]), randomInt);
    // Stop if old pTic[t] was larger
    if (old > randomInt) finished = 1;

    // Lock two extra triangles
    if ((t == tTarget || tPrev == tTarget) && finished == 0) {
      int t1 = pEt[eBound].x;
      if (t1 == t) t1 = pEt[eBound].y;

      if (t1 != -1) {
        // Set pTiC to maximum of pTiC and pRandom
        int old = AtomicMax(&(pTriangleLock[t1]), randomInt);
        // Stop if old pTic[t] was larger
        if (old > randomInt) finished = 1;
      }
    }

    tPrev = t;
    int tNext = pEt[eCross].x;
    if (tNext == t) tNext = pEt[eCross].y;
    t = tNext;

    if (t == tStart || t == -1) finished = 1;
  }
}

//#########################################################################
//#########################################################################

__global__ void
devLockTriangles(int nRemove,
                 int *pVertexRemove, int *pVertexTriangle,
                 int3 *pTv, int3 *pTe, int2 *pEt,
                 int nVertex, int *pTriangleTarget,
                 unsigned int *pRandom, int *pTriangleLock)
{
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nRemove) {
    LockTrianglesSingle(pVertexRemove[i],
                        pVertexTriangle[i],
                        pTv, pTe, pEt,
                        nVertex, pTriangleTarget[i],
                        pRandom[i], pTriangleLock);

    i += gridDim.x*blockDim.x;
  }
}

//#########################################################################
//#########################################################################

void Coarsen::LockTriangles(Connectivity *connectivity,
                            Array<int> *triangleLock)
{
  triangleLock->SetToValue(-1);

  int nRemove = vertexRemove->GetSize();
  int nVertex = connectivity->vertexCoordinates->GetSize();
  int nTriangle = connectivity->triangleVertices->GetSize();

  int *pVertexRemove = vertexRemove->GetPointer();
  int *pTriangleTarget = triangleTarget->GetPointer();
  int *pVertexTriangle = vertexTriangle->GetPointer();

  int *pTriangleLock = triangleLock->GetPointer();

  int3 *pTv = connectivity->triangleVertices->GetPointer();
  int3 *pTe = connectivity->triangleEdges->GetPointer();
  int2 *pEt = connectivity->edgeTriangles->GetPointer();

  // Shuffle points to add to maximise parallelisation
  unsigned int *pRandom = randomUnique->GetPointer();

  // Adjust state
  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devLockTriangles,
                                       (size_t) 0, 0);

    devLockTriangles<<<nBlocks, nThreads>>>
      (nRemove, pVertexRemove,
       pVertexTriangle,
       pTv, pTe, pEt,
       nVertex, pTriangleTarget,
       pRandom, pTriangleLock);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int n = 0; n < nRemove; n++)
      LockTrianglesSingle(pVertexRemove[n],
                          pVertexTriangle[n],
                          pTv, pTe, pEt,
                          nVertex, pTriangleTarget[n],
                          pRandom[n], pTriangleLock);
  }

}


}  // namespace astrix
