#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file checkedge.cu
\brief Functions for checking edges in Mesh for Delaunay property*/
#include <iostream>
//#include <fstream>

#include "../../Common/definitions.h"
#include "../../Array/array.h"
#include "../Predicates/predicates.h"
#include "./delaunay.h"
#include "./../triangleLow.h"
#include "../../Common/cudaLow.h"
#include "../Connectivity/connectivity.h"
#include "../Param/meshparameter.h"
#include "../../Common/profile.h"

namespace astrix {
    
//#########################################################################
/*! \brief Check edge \a i for Delaunay-hood

Check edge \a i and write result in \a eNonDel (1 if not Delaunay, 0 otherwise)

\param i Index of edge to check
\param *pVc Pointer to vertex coordinates
\param *pTv Pointer to triangle vertices
\param *pTe Pointer to triangle edges
\param *pEt Pointer to edge triangles
\param *pEnd Pointer to list of edges that are not Delaunay (output)
\param *pred Pointer to initialised Predicates object
\param *pParam Pointer to initialised Predicates parameter vector
\param nVertex Total number of vertices in Mesh
\param Px Periodic domain size x
\param Py Periodic domain size y*/
//#########################################################################

__host__ __device__
void CheckEdge(int i,
	       const real2* __restrict__ pVc,
	       const int3* __restrict__ pTv,
	       const int3* __restrict__ pTe,
	       const int2* __restrict__ pEt,
	       int *pEnd, const Predicates *pred, real *pParam,
	       int nVertex, real Px, real Py)
{
  // Assume edge is Delaunay
  int ret = -1;

  int t1 = pEt[i].x;
  int t2 = pEt[i].y;
  
  if (t1 != -1 && t2 != -1) {
    int a = pTv[t1].x;
    int b = pTv[t1].y;
    int c = pTv[t1].z;    

    int e1 = pTe[t1].x;
    int e2 = pTe[t1].y;
    int e3 = pTe[t1].z;

    int f =   (i == e1)*b +  (i == e2)*c +  (i == e3)*a;

    int d = (i == e1)*c + (i == e2)*a + (i == e3)*b;
    real dx, dy;
    GetTriangleCoordinatesSingle(pVc, d, nVertex, Px, Py, dx, dy);
    
    a = pTv[t2].x;
    b = pTv[t2].y;
    c = pTv[t2].z;
    
    real ax, bx, cx, ay, by, cy;
    GetTriangleCoordinates(pVc, a, b, c,
			   nVertex, Px, Py, 
			   ax, bx, cx, ay, by, cy);

    // Going to test if d lies in circle of t2
    e1 = pTe[t2].x;
    e2 = pTe[t2].y;
    e3 = pTe[t2].z;
    
    b = (i == e1)*a + (i == e2)*b + (i == e3)*c;

    // Edge is between (e, c) and (f, b)
    
    // PERIODIC
    TranslateVertexToVertex(b, f, Px, Py, nVertex, dx, dy);

    real detNew = pred->incircle(ax, ay, bx, by, cx, cy, dx, dy, pParam);

    // Edge is not Delaunay
    if (detNew > (real) 0.0) ret = i;
  }

  pEnd[i] = ret;
}
  
//######################################################################
/*! \brief Kernel checking edges for Delaunay-hood

Check edges and write result in \a pEnd (1 if not Delaunay, 0 otherwise)

\param nEdge Total number of edges in Mesh
\param *pVc Pointer to vertex coordinates
\param *pTv Pointer to triangle vertices
\param *pTe Pointer to triangle edges
\param *pEt Pointer to edge triangles
\param *pEnd Pointer to list of edges that are not Delaunay (output)
\param *pred Pointer to initialised Predicates object
\param *pParam Pointer to initialised Predicates parameter vector
\param nVertex Total number of vertices in Mesh
\param Px Periodic domain size x
\param Py Periodic domain size y*/
//######################################################################

__global__ void
devCheckEdge(int nEdge,
	     const real2* __restrict__ pVc,
	     const int3* __restrict__ pTv,
	     const int3* __restrict__ pTe,
	     const int2* __restrict__ pEt,
	     int *pEnd, const Predicates *pred, real *pParam,
	     int nVertex, real Px, real Py)
{
  // i = edge number
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nEdge) {
    CheckEdge(i, pVc, pTv, pTe, pEt, pEnd, pred, pParam, nVertex, Px, Py);

    // Next edge
    i += blockDim.x*gridDim.x;
  }
}

__global__ void
devCheckEdgeLimit(int nEdgeCheck,
		  const int *pEnC,
		  const real2* __restrict__ pVc,
		  const int3* __restrict__ pTv,
		  const int3* __restrict__ pTe,
		  const int2* __restrict__ pEt,
		  int *pEnd, const Predicates *pred, real *pParam,
		  int nVertex, real Px, real Py)
{
  // i = edge number
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nEdgeCheck) {
    int e = pEnC[i];
    CheckEdge(e, pVc, pTv, pTe, pEt, pEnd, pred, pParam, nVertex, Px, Py);

    // Next edge
    i += blockDim.x*gridDim.x;
  }
}

//#########################################################################
/*! Check edges for Delaunay-hood. Result is written in \a edgeNonDelaunay (-1 if Delaunay)

\param *connectivity Pointer to basic Mesh data
\param *predicates Pointer to exact geometric predicates
\param *meshParameter Pointer to Mesh parameters*/
//#########################################################################

void Delaunay::CheckEdges(Connectivity * const connectivity,
			  const Predicates *predicates,
			  const MeshParameter *meshParameter,
			  Array<int> * const edgeNeedsChecking,
			  const int nEdgeCheck)
{
  int nVertex = connectivity->vertexCoordinates->GetSize();
  int nEdge = connectivity->edgeTriangles->GetSize();
  
#ifdef TIME_ASTRIX
  hipEvent_t start, stop;
  float elapsedTime = 0.0f;
  hipEventCreate(&start);
  hipEventCreate(&stop);
#endif

  real2 *pVc = connectivity->vertexCoordinates->GetPointer();
  int3 *pTv = connectivity->triangleVertices->GetPointer();
  int3 *pTe = connectivity->triangleEdges->GetPointer();
  int2 *pEt = connectivity->edgeTriangles->GetPointer();
 
  int *pEnd = edgeNonDelaunay->GetPointer();
  
  real *pParam = predicates->GetParamPointer(cudaFlag);

  real Px = meshParameter->maxx - meshParameter->minx;
  real Py = meshParameter->maxy - meshParameter->miny;

  if (edgeNeedsChecking == 0) {
    if (cudaFlag == 1) {
      int nBlocks = 128;
      int nThreads = 128;
      
      // Base nThreads and nBlocks on maximum occupancy
      hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
					 devCheckEdge, 
					 (size_t) 0, 0);
      
#ifdef TIME_ASTRIX
      hipEventRecord(start, 0);
#endif
      
      devCheckEdge<<<nBlocks, nThreads>>>
	(nEdge, pVc, pTv, pTe, pEt, pEnd, predicates, pParam, nVertex, Px, Py);
      
#ifdef TIME_ASTRIX
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
#endif
      
      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );
    } else {
#ifdef TIME_ASTRIX
      hipEventRecord(start, 0);
#endif

      for (int i = 0; i < nEdge; i++) 
	CheckEdge(i, pVc, pTv, pTe, pEt, pEnd, predicates,
		  pParam, nVertex, Px, Py);

      // Make structured mesh less uniform
      if (meshParameter->structuredFlag == 2) {
	real Px = meshParameter->maxx - meshParameter->minx;
	real Py = meshParameter->maxy - meshParameter->miny;
      
	int nx = (int) (sqrt(0.565/meshParameter->baseResolution)*Px) + 4;
	int ny = (int)(nx*Py/Px);
	for (int i = 0; i < nx - 1; i += 2) {
	  for (int j = 0; j < ny - 1; j += 3) {
	    int v = j*(nx - meshParameter->periodicFlagX) + i;
	    int e = 3*(v - j) + j + 1 + 2*j*meshParameter->periodicFlagX -
	      (i > 0)*meshParameter->periodicFlagX;

	    pEnd[e] = e;
	  }
	}
      }
      

#ifdef TIME_ASTRIX
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
#endif
    }
  } else {
    int *pEnC = edgeNeedsChecking->GetPointer();
    edgeNonDelaunay->SetToValue(-1);
    
    if (cudaFlag == 1) {
      int nBlocks = 128;
      int nThreads = 128;
      
      // Base nThreads and nBlocks on maximum occupancy
      hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
					 devCheckEdgeLimit, 
					 (size_t) 0, 0);
      
#ifdef TIME_ASTRIX
      hipEventRecord(start, 0);
#endif
      
      devCheckEdgeLimit<<<nBlocks, nThreads>>>
	(nEdgeCheck, pEnC, pVc, pTv, pTe, pEt, pEnd,
	 predicates, pParam, nVertex, Px, Py);
      
#ifdef TIME_ASTRIX
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
#endif
      
      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );
    } else {
#ifdef TIME_ASTRIX
      hipEventRecord(start, 0);
#endif

      for (int i = 0; i < nEdgeCheck; i++) 
	CheckEdge(pEnC[i], pVc, pTv, pTe, pEt, pEnd, predicates,
		  pParam, nVertex, Px, Py);
      
#ifdef TIME_ASTRIX
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
#endif
    }
  }
   
#ifdef TIME_ASTRIX
  hipEventElapsedTime(&elapsedTime, start, stop);
  WriteProfileFile("CheckEdge.txt", nEdgeCheck, elapsedTime, cudaFlag);
#endif

}
  
}
