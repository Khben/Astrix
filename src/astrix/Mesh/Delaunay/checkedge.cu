#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file checkedge.cu
\brief Functions for checking edges in Mesh for Delaunay property

\section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/
#include <iostream>

#include "../../Common/definitions.h"
#include "../../Array/array.h"
#include "../Predicates/predicates.h"
#include "./delaunay.h"
#include "./../triangleLow.h"
#include "../../Common/cudaLow.h"
#include "../Connectivity/connectivity.h"
#include "../Param/meshparameter.h"
#include "../../Common/profile.h"

namespace astrix {

//#########################################################################
/*! \brief Check edge \a i for Delaunay-hood

Check edge \a i and write result in \a eNonDel (1 if not Delaunay, 0 otherwise)

\param i Index of edge to check
\param *pVc Pointer to vertex coordinates
\param *pTv Pointer to triangle vertices
\param *pTe Pointer to triangle edges
\param *pEt Pointer to edge triangles
\param *pEnd Pointer to list of edges that are not Delaunay (output)
\param *pred Pointer to initialised Predicates object
\param *pParam Pointer to initialised Predicates parameter vector
\param nVertex Total number of vertices in Mesh
\param Px Periodic domain size x
\param Py Periodic domain size y*/
//#########################################################################

__host__ __device__
int CheckEdge(int i,
              real2 *pVc,
              const int3* __restrict__ pTv,
              const int3* __restrict__ pTe,
              int2 *pEt,
              const Predicates *pred, real *pParam,
              int nVertex, real Px, real Py)
{
  // Assume edge is Delaunay
  int ret = -1;

  int t1 = pEt[i].x;
  int t2 = pEt[i].y;

  if (t1 != -1 && t2 != -1) {
    int a = pTv[t1].x;
    int b = pTv[t1].y;
    int c = pTv[t1].z;

    int e1 = pTe[t1].x;
    int e2 = pTe[t1].y;
    int e3 = pTe[t1].z;

    int f =   (i == e1)*b +  (i == e2)*c +  (i == e3)*a;

    int d = (i == e1)*c + (i == e2)*a + (i == e3)*b;
    real dx, dy;
    GetTriangleCoordinatesSingle(pVc, d, nVertex, Px, Py, dx, dy);

    a = pTv[t2].x;
    b = pTv[t2].y;
    c = pTv[t2].z;

    real ax, bx, cx, ay, by, cy;
    GetTriangleCoordinates(pVc, a, b, c,
                           nVertex, Px, Py,
                           ax, bx, cx, ay, by, cy);

    // Going to test if d lies in circle of t2
    e1 = pTe[t2].x;
    e2 = pTe[t2].y;
    e3 = pTe[t2].z;

    b = (i == e1)*a + (i == e2)*b + (i == e3)*c;

    // Edge is between (e, c) and (f, b)

    // PERIODIC
    TranslateVertexToVertex(b, f, Px, Py, nVertex, dx, dy);

    real detNew = pred->incircle(ax, ay, bx, by, cx, cy, dx, dy, pParam);

    // Edge is not Delaunay
    if (detNew > (real) 0.0) ret = i;
  }

  return ret;
}

//######################################################################
/*! \brief Kernel checking edges for Delaunay-hood

Check edges and write result in \a pEnd (1 if not Delaunay, 0 otherwise)

\param nEdge Total number of edges in Mesh
\param *pVc Pointer to vertex coordinates
\param *pTv Pointer to triangle vertices
\param *pTe Pointer to triangle edges
\param *pEt Pointer to edge triangles
\param *pEnd Pointer to list of edges that are not Delaunay (output)
\param *pred Pointer to initialised Predicates object
\param *pParam Pointer to initialised Predicates parameter vector
\param nVertex Total number of vertices in Mesh
\param Px Periodic domain size x
\param Py Periodic domain size y*/
//######################################################################

__global__ void
devCheckEdge(int nEdge,
             real2 *pVc,
             const int3* __restrict__ pTv,
             const int3* __restrict__ pTe,
             int2 *pEt,
             int *pEnd, const Predicates *pred, real *pParam,
             int nVertex, real Px, real Py)
{
  // i = edge number
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nEdge) {
    pEnd[i] =
      CheckEdge(i, pVc, pTv, pTe, pEt, pred, pParam, nVertex, Px, Py);

    // Next edge
    i += blockDim.x*gridDim.x;
  }
}

//######################################################################
/*! \brief Kernel checking limited amount of edges for Delaunay-hood

Check edges and write result in \a pEnd (1 if not Delaunay, 0 otherwise). Only the first \a nEdgeCheck entries of \a pEnc are checked.

\param nEdgeCheck Total number of edges to check
\param *pEnC Pointer to array listing of edges to be checked
\param *pVc Pointer to vertex coordinates
\param *pTv Pointer to triangle vertices
\param *pTe Pointer to triangle edges
\param *pEt Pointer to edge triangles
\param *pEnd Pointer to list of edges that are not Delaunay (output)
\param *pred Pointer to initialised Predicates object
\param *pParam Pointer to initialised Predicates parameter vector
\param nVertex Total number of vertices in Mesh
\param Px Periodic domain size x
\param Py Periodic domain size y*/
//######################################################################

__global__ void
devCheckEdgeLimit(int nEdgeCheck,
                  int *pEnC,
                  real2 *pVc,
                  const int3* __restrict__ pTv,
                  const int3* __restrict__ pTe,
                  int2 *pEt,
                  int *pEnd, const Predicates *pred, real *pParam,
                  int nVertex, real Px, real Py)
{
  // i = edge number
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nEdgeCheck) {
    int e = pEnC[i];
    pEnd[i] =
      CheckEdge(e, pVc, pTv, pTe, pEt, pred, pParam, nVertex, Px, Py);

    // Next edge
    i += blockDim.x*gridDim.x;
  }
}

//#########################################################################
/*! Check edges for Delaunay-hood. Result is written in \a edgeNonDelaunay (-1 if Delaunay)

\param *connectivity Pointer to basic Mesh data
\param *predicates Pointer to exact geometric predicates
\param *meshParameter Pointer to Mesh parameters
\param *edgeNeedsChecking pEdgeNeedsChecking[i] = i if edge \a i needs to be checked. If \a edgeNeedsChecking == 0 then all edges are checked
\param nEdgeCheck Number of edges to check if \a edgeNeedsChecking != 0
*/
//#########################################################################

void Delaunay::CheckEdges(Connectivity * const connectivity,
                          const Predicates *predicates,
                          const MeshParameter *meshParameter,
                          Array<int> * const edgeNeedsChecking,
                          const int nEdgeCheck)
{
  int nVertex = connectivity->vertexCoordinates->GetSize();
  int nEdge = connectivity->edgeTriangles->GetSize();

#ifdef TIME_ASTRIX
  hipEvent_t start, stop;
  float elapsedTime = 0.0f;
  gpuErrchk( hipEventCreate(&start) );
  gpuErrchk( hipEventCreate(&stop) );
#endif

  real2 *pVc = connectivity->vertexCoordinates->GetPointer();
  int3 *pTv = connectivity->triangleVertices->GetPointer();
  int3 *pTe = connectivity->triangleEdges->GetPointer();
  int2 *pEt = connectivity->edgeTriangles->GetPointer();

  int *pEnd = edgeNonDelaunay->GetPointer();

  real *pParam = predicates->GetParamPointer(cudaFlag);

  real Px = meshParameter->maxx - meshParameter->minx;
  real Py = meshParameter->maxy - meshParameter->miny;

  if (edgeNeedsChecking == 0) {
    if (cudaFlag == 1) {
      int nBlocks = 128;
      int nThreads = 128;

      // Base nThreads and nBlocks on maximum occupancy
      hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                         devCheckEdge,
                                         (size_t) 0, 0);

#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(start, 0) );
#endif

      devCheckEdge<<<nBlocks, nThreads>>>
        (nEdge, pVc, pTv, pTe, pEt, pEnd, predicates, pParam, nVertex, Px, Py);

#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(stop, 0) );
      gpuErrchk( hipEventSynchronize(stop) );
#endif

      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );
    } else {
#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(start, 0) );
#endif

      for (int i = 0; i < nEdge; i++)
        pEnd[i] =
          CheckEdge(i, pVc, pTv, pTe, pEt, predicates,
                    pParam, nVertex, Px, Py);

      // Make structured mesh less uniform
      if (meshParameter->structuredFlag == 2) {
        real Px = meshParameter->maxx - meshParameter->minx;
        real Py = meshParameter->maxy - meshParameter->miny;

        int nx = (int) (sqrt(0.565/meshParameter->baseResolution)*Px) + 4;
        int ny = (int)(nx*Py/Px);
        for (int i = 0; i < nx - 1; i += 2) {
          for (int j = 0; j < ny - 1; j += 3) {
            int v = j*(nx - meshParameter->periodicFlagX) + i;
            int e = 3*(v - j) + j + 1 + 2*j*meshParameter->periodicFlagX -
              (i > 0)*meshParameter->periodicFlagX;

            pEnd[e] = e;
          }
        }
      }


#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(stop, 0) );
      gpuErrchk( hipEventSynchronize(stop) );
#endif
    }
  } else {
    int *pEnC = edgeNeedsChecking->GetPointer();
    edgeNonDelaunay->SetToValue(-1);

    if (cudaFlag == 1) {
      int nBlocks = 128;
      int nThreads = 128;

      // Base nThreads and nBlocks on maximum occupancy
      hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                         devCheckEdgeLimit,
                                         (size_t) 0, 0);

#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(start, 0) );
#endif

      devCheckEdgeLimit<<<nBlocks, nThreads>>>
        (nEdgeCheck, pEnC, pVc, pTv, pTe, pEt, pEnd,
         predicates, pParam, nVertex, Px, Py);

#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(stop, 0) );
      gpuErrchk( hipEventSynchronize(stop) );
#endif

      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );
    } else {
#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(start, 0) );
#endif

      for (int i = 0; i < nEdgeCheck; i++)
        pEnd[i] =
          CheckEdge(pEnC[i], pVc, pTv, pTe, pEt, predicates,
                    pParam, nVertex, Px, Py);

#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(stop, 0) );
      gpuErrchk( hipEventSynchronize(stop) );
#endif
    }
  }

#ifdef TIME_ASTRIX
  gpuErrchk( hipEventElapsedTime(&elapsedTime, start, stop) );
  WriteProfileFile("CheckEdge.prof", nEdgeCheck, elapsedTime, cudaFlag);
#endif
}

}  // namespace astrix
