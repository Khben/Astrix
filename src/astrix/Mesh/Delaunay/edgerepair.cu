#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file edgerepair.cu
\brief Functions for repairing edges in Mesh after flipping*/
#include <iostream>

#include "../../Common/definitions.h"
#include "../../Array/array.h"
#include "./delaunay.h"
#include "../../Common/cudaLow.h"
#include "../Connectivity/connectivity.h"
#include "../../Common/profile.h"

#include "../triangleLow.h"

namespace astrix {
  
//#########################################################################
/*! \brief Repair edge \a i if necessary

Check if edge is part of both neighbouring triangles. If not, it has been corrupted by a flip and we have to use \a tSub

\param i Edge index to consider
\param *pTsub Pointer to array of substitution triangles
\param *pTe Pointer to triangle edges
\param *pEt Pointer to edge triangles (modified)*/ 
//#########################################################################

__host__ __device__
void SingleEdgeRepair(int i, const int* __restrict__ pTsub,
		      const int3* __restrict__ pTe, int2 *pEt,
		      int printFlag)
{
  // Indices of triangles
  int t1 = pEt[i].x;
  int t2 = pEt[i].y;

  if (t1 != -1) {
    int tS1 = pTsub[t1];

    int e1 = pTe[t1].x;
    int e2 = pTe[t1].y;
    int e3 = pTe[t1].z;
    
    if (i != e1 && i != e2 && i != e3) {
      /*
#ifndef __CUDA_ARCH__
      if (printFlag == 1) {
	std::cout << "Error in edgeRepair!" << std::endl;
	std::cout << "Edge: " << i << std::endl;
	std::cout << "Triangles " << t1 << " " << t2 << std::endl;
	
	int qq; std::cin >> qq;
      }
#endif
      */
      t1 = tS1;
    }
  }
  
  if (t2 != -1) {
    int tS2 = pTsub[t2];

    int e1 = pTe[t2].x;
    int e2 = pTe[t2].y;
    int e3 = pTe[t2].z;
    
    if (i != e1 && i != e2 && i != e3) {
      /*
#ifndef __CUDA_ARCH__
      if (printFlag == 1) {
	std::cout << "Error in edgeRepair!" << std::endl;
	std::cout << "Edge: " << i << std::endl;
	std::cout << "Triangles " << t1 << " " << t2 << std::endl;

	int qq; std::cin >> qq;
      }
#endif
      */
      t2 = tS2;
    }
  }
    
  pEt[i].x = t1;
  pEt[i].y = t2;
}
    
//#########################################################################
/*! \brief Kernel repair edges if necessary

Check if all edges are part of both their neighbouring triangles. If not, it has been corrupted by a flip and we have to use \a pTsub

\param nEdge Total number of edges in Mesh
\param *pTsub Pointer to array of substitution triangles
\param *pTe Pointer to triangle edges
\param *pEt Pointer to edge triangles (modified)*/ 
//#########################################################################

__global__ void
devEdgeRepair(int nEdge,
	      const int* __restrict__ pTsub,
	      const int3* __restrict__ pTe,
	      int2 *pEt)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  while (i < nEdge) {
    SingleEdgeRepair(i, pTsub, pTe, pEt, 1);

    i += gridDim.x*blockDim.x;
  }
}

//#########################################################################
/*! \brief Kernel repair limited amount of edges if necessary

Check if edges are part of both their neighbouring triangles. If not, it has been corrupted by a flip and we have to use \a pTsub. Only edges part of insertion cavities are checked.

\param nEdgeCheck Total number of edges to check
\param *pEnC Array listing edges to check
\param *pTsub Pointer to array of substitution triangles
\param *pTe Pointer to triangle edges
\param *pEt Pointer to edge triangles (modified)*/ 
//#########################################################################

__global__ void
devEdgeRepairLimit(int nEdgeCheck,
		   const int *pEnC,
		   const int* __restrict__ pTsub,
		   const int3* __restrict__ pTe,
		   int2 *pEt)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  while (i < nEdgeCheck) {
    int e = pEnC[i];
    SingleEdgeRepair(e, pTsub, pTe, pEt, 0);

    i += gridDim.x*blockDim.x;
  }
}

//#########################################################################
/*! Repair edges if necessary. An edge flip can corrupt the mesh, resulting in the edge having the wrong neighbouring triangles; fortunately, it is relatively easy to repair by using the previously computed \a triangleSubstitute Array.

\param *connectivity Pointer to basic Mesh data
\param *edgeNeedsChecking pEdgeNeedsChecking[i] = i if edge \a i was part of the insertion cavity of a new vertex. If \a edgeNeedsChecking == 0 then all edges are checked
\param nEdgeCheck Number of edges to check if \a edgeNeedsChecking != 0
*/
//#########################################################################

void Delaunay::EdgeRepair(Connectivity * const connectivity,
			  Array<int> * const edgeNeedsChecking,
			  const int nEdgeCheck)
{
  int nEdge = connectivity->edgeTriangles->GetSize();
  
#ifdef TIME_ASTRIX
  hipEvent_t start, stop;
  float elapsedTime = 0.0f;
  gpuErrchk( hipEventCreate(&start) ) ;
  gpuErrchk( hipEventCreate(&stop) );
#endif

  int3 *pTe = connectivity->triangleEdges->GetPointer();
  int2 *pEt = connectivity->edgeTriangles->GetPointer();

  int *pTsub = triangleSubstitute->GetPointer();
 
  if (edgeNeedsChecking == 0) {

    /*
    int nVertex = connectivity->vertexCoordinates->GetSize();
    int nTriangle = connectivity->triangleVertices->GetSize();
    real2 *pVc = connectivity->vertexCoordinates->GetPointer();
    int3 *pTv = connectivity->triangleVertices->GetPointer();
    real Px = 1.0;
    real Py = 1.0;
    int t = 23208870;
    if (nTriangle >= t) {
      int a = pTv[t].x;
      int b = pTv[t].y;
      int c = pTv[t].z;
      
      real ax, bx, cx, ay, by, cy;
      GetTriangleCoordinates(pVc, a, b, c, nVertex, Px, Py,
			     ax, bx, cx, ay, by, cy);
      std::cout << "Triangle " << t << " coordinates: "
		<< ax << " " << ay << " " << bx << " " << by << " "
		<< cx << " " << cy << std::endl;
      while (a >= nVertex) a -= nVertex;
      while (a < 0) a += nVertex;
      while (b >= nVertex) b -= nVertex;
      while (b < 0) b += nVertex;
      while (c >= nVertex) c -= nVertex;
      while (c < 0) c += nVertex;
      std::cout << "Triangle " << t << " vertices: "
		<< a << " " << b << " " << c << std::endl;
    }
    t = 5483649;
    if (nTriangle >= t) {
      int a = pTv[t].x;
      int b = pTv[t].y;
      int c = pTv[t].z;
      
      real ax, bx, cx, ay, by, cy;
      GetTriangleCoordinates(pVc, a, b, c, nVertex, Px, Py, 
			     ax, bx, cx, ay, by, cy);
      std::cout << "Triangle " << t << " coordinates: "
		<< ax << " " << ay << " " << bx << " " << by << " "
		<< cx << " " << cy << std::endl;
      while (a >= nVertex) a -= nVertex;
      while (a < 0) a += nVertex;
      while (b >= nVertex) b -= nVertex;
      while (b < 0) b += nVertex;
      while (c >= nVertex) c -= nVertex;
      while (c < 0) c += nVertex;
      std::cout << "Triangle " << t << " vertices: "
		<< a << " " << b << " " << c << std::endl;
    }
    */
    
    if (cudaFlag == 1) {
      int nBlocks = 128;
      int nThreads = 128; 
      
      // Base nThreads and nBlocks on maximum occupancy
      hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
					 devEdgeRepair, 
					 (size_t) 0, 0);

#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(start, 0) );
#endif
      devEdgeRepair<<<nBlocks, nThreads>>>
	(nEdge, pTsub, pTe, pEt);
#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(stop, 0) );
      gpuErrchk( hipEventSynchronize(stop) );
#endif
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
    } else {
#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(start, 0) );
#endif
      for (int i = 0; i < nEdge; i++) 
	SingleEdgeRepair(i, pTsub, pTe, pEt, 1);
#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(stop, 0) );
      gpuErrchk( hipEventSynchronize(stop) );
#endif
    }

#ifdef TIME_ASTRIX
  gpuErrchk( hipEventElapsedTime(&elapsedTime, start, stop) );
  WriteProfileFile("EdgeRepair.prof", nEdge, elapsedTime, cudaFlag);
#endif

  } else {

    int *pEnC = edgeNeedsChecking->GetPointer();

    if (cudaFlag == 1) {
      int nBlocks = 128;
      int nThreads = 128; 
      
      // Base nThreads and nBlocks on maximum occupancy
      hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
					 devEdgeRepairLimit, 
					 (size_t) 0, 0);

#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(start, 0) );
#endif
      devEdgeRepairLimit<<<nBlocks, nThreads>>>
	(nEdgeCheck, pEnC, pTsub, pTe, pEt);
#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(stop, 0) );
      gpuErrchk( hipEventSynchronize(stop) );
#endif
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
    } else {
#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(start, 0) );
#endif
      for (int i = 0; i < nEdgeCheck; i++) 
	SingleEdgeRepair(pEnC[i], pTsub, pTe, pEt, 0);
#ifdef TIME_ASTRIX
      gpuErrchk( hipEventRecord(stop, 0) );
      gpuErrchk( hipEventSynchronize(stop) );
#endif
    }
    
#ifdef TIME_ASTRIX
    gpuErrchk( hipEventElapsedTime(&elapsedTime, start, stop) );
    WriteProfileFile("EdgeRepair.prof", nEdgeCheck, elapsedTime, cudaFlag);
#endif
  }
  
}

}
