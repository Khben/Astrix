#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file vertexarea.cu
\brief Functions for calculating vertex areas

\section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/
#include <iostream>

#include "../../Common/definitions.h"
#include "../../Array/array.h"
#include "./delaunay.h"
#include "./../triangleLow.h"
#include "../../Common/atomic.h"
#include "../../Common/cudaLow.h"
#include "../Connectivity/connectivity.h"
#include "../Param/meshparameter.h"

namespace astrix {

//##############################################################################
/*! \brief Add contribution of area of triangle \a n to the area of its vertices

  Every triangle contributes one third of its area to the area of the Voronoi cell associated with its vertices. Atomically add this contribution to \a vertexArea

\param n Index of triangle to consider
\param *tv1 Pointer to first vertex of triangle
\param *tv2 Pointer to second vertex of triangle
\param *tv3 Pointer to third vertex of triangle
\param *pVertexArea Pointer to output array containing vertex areas
\param nVertex Total number of vertices in Mesh
\param *pVertX Pointer to x-coordinates of vertices
\param *pVertY Pointer to y-coordinates of vertices
\param Px Periodic domain size x
\param Py Periodic domain size y
\param *triL Pointer to array containing triangle edge lengths
\param nTriangle Total number of triangles*/
//##############################################################################

__host__ __device__
void VertexAreaSingle(int n, int3 *pTv,
                      real *pVertexArea, int nVertex,
                      real2 *pVc, real Px, real Py)
{
  const real onethird = (real) (1.0/3.0);
  const real half  = (real) 0.5;

  int a = pTv[n].x;
  int b = pTv[n].y;
  int c = pTv[n].z;

  real Ax, Bx, Cx, Ay, By, Cy;
  GetTriangleCoordinates(pVc, a, b, c,
                         nVertex, Px, Py,
                         Ax, Bx, Cx, Ay, By, Cy);

  real A = half*((Ax - Cx)*(By - Cy) - (Ay - Cy)*(Bx - Cx))*onethird;

  while (a >= nVertex) a -= nVertex;
  while (b >= nVertex) b -= nVertex;
  while (c >= nVertex) c -= nVertex;
  while (a < 0) a += nVertex;
  while (b < 0) b += nVertex;
  while (c < 0) c += nVertex;

  AtomicAdd(&pVertexArea[a], A);
  AtomicAdd(&pVertexArea[b], A);
  AtomicAdd(&pVertexArea[c], A);
}

//######################################################################
/*! \brief Kernel calculating vertex areas (Voronoi cells)

  Every triangle contributes one third of its area to the area of the Voronoi cell associated with its vertices. Atomically add this contribution to \a vertexArea

\param nVertex Total number of vertices in Mesh
\param nTriangle Total number of triangles
\param *tv1 Pointer to first vertex of triangle
\param *tv2 Pointer to second vertex of triangle
\param *tv3 Pointer to third vertex of triangle
\param *pVertexArea Pointer to output array containing vertex areas
\param *pVertX Pointer to x-coordinates of vertices
\param *pVertY Pointer to y-coordinates of vertices
\param Px Periodic domain size x
\param Py Periodic domain size y
\param *triL Pointer to array containing triangle edge lengths*/
//######################################################################

__global__ void
devDCalcVertexArea(int nVertex, int nTriangle,
                   int3 *pTv, real *pVertexArea,
                   real2 *pVc, real Px, real Py)
{
  // n = triangle number
  int n = blockIdx.x*blockDim.x + threadIdx.x;

  while (n < nTriangle) {
    VertexAreaSingle(n, pTv, pVertexArea, nVertex, pVc, Px, Py);

    n += blockDim.x*gridDim.x;
  }
}

//#########################################################################
/*! Every triangle contributes one third of its area to the area of the Voronoi cell associated with its vertices. Atomically add this contribution to \a vertexArea

\param *connectivity Pointer to basic Mesh data
\param *meshParameter Pointer to mesh parameters*/
//#########################################################################

void Delaunay::CalcVertexArea(Connectivity * const connectivity,
                              const MeshParameter *meshParameter)
{
  int nTriangle = connectivity->triangleVertices->GetSize();
  int nVertex = connectivity->vertexCoordinates->GetSize();

  real2 *pVc = connectivity->vertexCoordinates->GetPointer();
  int3 *pTv = connectivity->triangleVertices->GetPointer();

  real Px = meshParameter->maxx - meshParameter->minx;
  real Py = meshParameter->maxy - meshParameter->miny;

  // Vertex area (= volume Voronoi cell)
  vertexArea->SetSize(nVertex);
  vertexArea->SetToValue(0.0);
  real *pVertexArea = vertexArea->GetPointer();

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devDCalcVertexArea,
                                       (size_t) 0, 0);

    devDCalcVertexArea<<<nBlocks, nThreads>>>
      (nVertex, nTriangle, pTv, pVertexArea, pVc, Px, Py);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int n = 0; n < nTriangle; n++)
      VertexAreaSingle(n, pTv, pVertexArea, nVertex, pVc, Px, Py);
  }
}

}
