#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file findparallelflip.cu
\brief File containing function to find parallel flip set.

*/ /* \section LICENSE
Copyright (c) 2017 Sijme-Jan Paardekooper

This file is part of Astrix.

Astrix is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

Astrix is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Astrix.  If not, see <http://www.gnu.org/licenses/>.*/
#include <iostream>

#include "../../Common/definitions.h"
#include "../../Array/array.h"
#include "./delaunay.h"
#include "../../Common/cudaLow.h"
#include "../Connectivity/connectivity.h"
#include "../../Common/atomic.h"
#include "../../Common/profile.h"

namespace astrix {

//############################################################################
/*! \brief Kernel filling array with triangles that are affected by flipping edge

  Triangles affected are those adjacent to the edge to be flipped

\param nFlip Number of edges to be flipped
\param *pTaff Output array containing affected triangles
\param *pTaffEdge Output array containing the index in \a pEnd of the edge to be flipped
\param *pEnd Pointer to array containing edges to be flipped
\param *pEt Pointer to edge triangles*/
//############################################################################

__global__ void
devFillAffectedTriangles(int nFlip, int *pTaff, int *pTaffEdge,
                         int *pEnd, int2 *pEt)
{
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nFlip) {
    int e = pEnd[i];

    pTaffEdge[i] = i;
    pTaffEdge[i + nFlip] = i;

    pTaff[i]         = pEt[e].x;
    pTaff[i + nFlip] = pEt[e].y;

    i += gridDim.x*blockDim.x;
  }
}

//############################################################################
/*! \brief Fill array with triangles that are affected by flipping edge

  Triangles affected are those adjacent to the edge to be flipped

\param *triangleAffected Output Array containing affected triangles
\param *triangleAffectedEdge Output Array containing the index in \a pEnd of the edge to be flipped
\param *edgeNonDelaunay Pointer to Array containing edges to be flipped
\param *connectivity Pointer to basic Mesh data
\param nFlip Number of edges to be flipped
\param cudaFlag Flag whether to do computation on device. This has to match the parameter cudaFlag of the Arrays*/
//############################################################################

void FillAffectedTriangles(Array<int> * const triangleAffected,
                           Array<int> * const triangleAffectedEdge,
                           const Array<int> *edgeNonDelaunay,
                           Connectivity * const connectivity,
                           const int nFlip,
                           const int cudaFlag)
{
  int2 *pEt = connectivity->edgeTriangles->GetPointer();
  int *pEnd = edgeNonDelaunay->GetPointer();

  int *pTaff = triangleAffected->GetPointer();
  int *pTaffEdge = triangleAffectedEdge->GetPointer();

  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
                                       devFillAffectedTriangles,
                                       (size_t) 0, 0);

    devFillAffectedTriangles<<<nBlocks, nThreads>>>
      (nFlip, pTaff, pTaffEdge, pEnd, pEt);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int i = 0; i < nFlip; i++) {
      int e = pEnd[i];

      pTaffEdge[i] = i;
      pTaffEdge[i + nFlip] = i;

      pTaff[i]         = pEt[e].x;
      pTaff[i + nFlip] = pEt[e].y;
    }
  }
}

//#############################################################################
/*! Compact the Array \a edgeNonDelaunay into a set of edges that can be flipped in parallel

\param *connectivity Pointer to basic Mesh data
\param nFlip Number of edges that are not Delaunay*/
//#############################################################################

int Delaunay::FindParallelFlipSet(Connectivity * const connectivity,
                                  const int nFlip)
{
  // Fill triangleAffected and triangleAffectedEdge (direct only)
  FillAffectedTriangles(triangleAffected,
                        triangleAffectedEdge,
                        edgeNonDelaunay,
                        connectivity,
                        nFlip, cudaFlag);

  int firstEdge;
  edgeNonDelaunay->GetSingleValue(&firstEdge, 0);

  // Sort triangleAffected; reindex triangleAffectedEdge
  triangleAffected->SortByKey(triangleAffectedEdge, 2*nFlip);

  // Set edgeNonDelaunay[i] = -1 for non-unique triangles
  edgeNonDelaunay->ScatterUnique(triangleAffected, triangleAffectedEdge,
                                 2*nFlip, -1, -1);

  // Keep only entries >= 0 (note: size of Array not changed!)
  int nFlipParallel = edgeNonDelaunay->RemoveValue(-1, nFlip);

  // Pathological case
  if (nFlipParallel == 0 && nFlip > 0) {
    edgeNonDelaunay->SetSingleValue(firstEdge, 0);
    nFlipParallel = 1;
  }

  return nFlipParallel;
}

}  // namespace astrix
