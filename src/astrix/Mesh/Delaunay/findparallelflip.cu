#include "hip/hip_runtime.h"
// -*-c++-*-
/*! \file findparallelflip.cu
\brief File containing function to find parallel flip set.*/

#include "../../Common/definitions.h"
#include "../../Array/array.h"
#include "./delaunay.h"
#include "../../Common/cudaLow.h"
#include "../Connectivity/connectivity.h"
#include "../../Common/atomic.h"

namespace astrix {

__host__ __device__
void SelectParallelFlip(int i, int *pEdgeNonDelaunay,
			int *pTriangleTaken,
			const int2* __restrict__ pEt)
{
  int canBeFlipped = 0;
  int e = pEdgeNonDelaunay[i];
  int t1 = pEt[e].x;
  int t2 = pEt[e].y;

  int tTaken1 = 0;
  if (t1 != -1) tTaken1 = AtomicCAS(&(pTriangleTaken[t1]), 0, 1);
  if (tTaken1 == 0) {
    int tTaken2 = 0;
    if (t2 != -1) tTaken2 = AtomicCAS(&(pTriangleTaken[t2]), 0, 1);
    if (tTaken2 == 0) canBeFlipped = 1;
    else if (t1 != -1) pTriangleTaken[t1] = 0;
  }
  
  if (canBeFlipped == 0) e = -1;
  pEdgeNonDelaunay[i] = e;
}

__global__ void 
devSelectParallelFlip(int nFlip,
		      int *pEdgeNonDelaunay,
		      int *pTriangleTaken,
		      const int2* __restrict__ pEt)
{
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nFlip) {
    SelectParallelFlip(i, pEdgeNonDelaunay, pTriangleTaken, pEt);
    
    i += gridDim.x*blockDim.x;
  }
}

  
//############################################################################
/*! \brief Kernel filling array with triangles that are affected by flipping edge

  Triangles affected are those adjacent to the edge to be flipped

\param nFlip Number of edges to be flipped
\param *pTaff Output array containing affected triangles
\param *pTaffEdge Output array containing the index in \a pEnd of the edge to be flipped
\param *pEnd Pointer to array containing edges to be flipped
\param *pEt Pointer to edge triangles*/
//############################################################################

__global__ void 
devFillAffectedTriangles(int nFlip, int *pTaff, int *pTaffEdge,
			 int *pEnd, int2 *pEt)
{
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  while (i < nFlip) {
    int e = pEnd[i];
    
    pTaffEdge[i] = i;
    pTaffEdge[i + nFlip] = i;

    pTaff[i]         = pEt[e].x;
    pTaff[i + nFlip] = pEt[e].y;
    
    i += gridDim.x*blockDim.x;
  }
}
  
//############################################################################
/*! \brief Fill array with triangles that are affected by flipping edge

  Triangles affected are those adjacent to the edge to be flipped

\param nFlip Number of edges to be flipped
\param *triangleAffected Output Array containing affected triangles
\param *triangleAffectedEdge Output Array containing the index in \a pEnd of the edge to be flipped
\param *edgeNonDelaunay Pointer to Array containing edges to be flipped
\param *edgeTriangles Pointer to Array containing neighbouring triangles to edges
\param nFlip Number of edges to be flipped
\param cudaFlag Flag whether to do computation on device. This has to match the parameter cudaFlag of the Arrays*/
//############################################################################

void FillAffectedTriangles(Array<int> * const triangleAffected,
			   Array<int> * const triangleAffectedEdge,
			   const Array<int> *edgeNonDelaunay,
			   Connectivity * const connectivity,
			   const int nFlip,
			   const int cudaFlag)
{
  int2 *pEt = connectivity->edgeTriangles->GetPointer();
  int *pEnd = edgeNonDelaunay->GetPointer();

  int *pTaff = triangleAffected->GetPointer();
  int *pTaffEdge = triangleAffectedEdge->GetPointer();

 if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128; 

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
				       devFillAffectedTriangles, 
				       (size_t) 0, 0);

    devFillAffectedTriangles<<<nBlocks, nThreads>>>
      (nFlip, pTaff, pTaffEdge, pEnd, pEt);
    
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int i = 0; i < nFlip; i++) {
      int e = pEnd[i];
      
      pTaffEdge[i] = i;
      pTaffEdge[i + nFlip] = i;

      pTaff[i]         = pEt[e].x;
      pTaff[i + nFlip] = pEt[e].y;
    }
  }
}

//#############################################################################
/*! Compact the Array \a edgeNonDelaunay into a set of edges that can be flipped in parallel

\param *connectivity Pointer to basic Mesh data
\param nFlip Number of edges that are not Delaunay*/
//#############################################################################

int Delaunay::FindParallelFlipSet(Connectivity * const connectivity,
				  const int nFlip)
{
  /*
  int nTriangle = connectivity->triangleVertices->GetSize();
  
  Array<int> *triangleTaken = new Array<int>(1, cudaFlag, nTriangle);
  triangleTaken->SetToValue(0);
  int *pTriangleTaken = triangleTaken->GetPointer();
  int *pEdgeNonDelaunay = edgeNonDelaunay->GetPointer();

  int2 *pEt = connectivity->edgeTriangles->GetPointer();
  
  if (cudaFlag == 1) {
    int nBlocks = 128;
    int nThreads = 128;

    // Base nThreads and nBlocks on maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads,
				       devSelectParallelFlip, 
				       (size_t) 0, 0);

    devSelectParallelFlip<<<nBlocks, nThreads>>>
      (nFlip, pEdgeNonDelaunay, pTriangleTaken, pEt);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  } else {
    for (int i = 0; i < nFlip; i++) 
      SelectParallelFlip(i, pEdgeNonDelaunay, pTriangleTaken, pEt);
  }
  
  // Keep only entries >= 0 (note: size of Array not changed!)
  int nFlipParallel = edgeNonDelaunay->RemoveValue(-1, nFlip);

  delete triangleTaken;
  */
  
  // Fill triangleAffected and triangleAffectedEdge (direct only)
  FillAffectedTriangles(triangleAffected,
			triangleAffectedEdge,
			edgeNonDelaunay,
			connectivity,
			nFlip, cudaFlag);

  int firstEdge;
  edgeNonDelaunay->GetSingleValue(&firstEdge, 0);
  
  // Sort triangleAffected; reindex triangleAffectedEdge
  triangleAffected->SortByKey(triangleAffectedEdge, 2*nFlip);

  // Set edgeNonDelaunay[i] = -1 for non-unique triangles
  edgeNonDelaunay->ScatterUnique(triangleAffected, triangleAffectedEdge,
				 2*nFlip, -1, -1);

  // Keep only entries >= 0 (note: size of Array not changed!)
  int nFlipParallel = edgeNonDelaunay->RemoveValue(-1, nFlip);
  
  // Pathological case
  if (nFlipParallel == 0 && nFlip > 0) {
    edgeNonDelaunay->SetSingleValue(firstEdge, 0);
    nFlipParallel = 1;
  }
  
  //std::cout << nFlip << " " << nFlipParallel << std::endl;
  
  
  return nFlipParallel;
}

}
